#include "hip/hip_runtime.h"
#include "KernelInterface.hpp"

void KernelInterface::runInitKernel(DistributionFunction* df,
                                    DistributionFunction* dfT,
                                    Partition partition,
                                    float rho,
                                    float vx,
                                    float vy,
                                    float vz,
                                    float T) {
  float sq_term = -1.5f * (vx * vx + vy * vy + vz * vz);
  Eigen::Vector3i n = partition.getArrayExtents();
  dim3 gridSize(n.y(), n.z(), 1);
  dim3 blockSize(n.x(), 1, 1);
  real* dfPtr = df->gpu_ptr(partition);
  real* dfTPtr = dfT->gpu_ptr(partition);

  InitKernel<<<gridSize, blockSize>>>(dfPtr, dfTPtr, n.x(), n.y(), n.z(), rho,
                                      vx, vy, vz, T, sq_term);

  CUDA_CHECK_ERRORS("InitKernel");
}

void KernelInterface::runComputeKernelInterior(
    const Partition partition,
    ComputeParams* par,
    DisplayQuantity::Enum displayQuantity,
    hipStream_t stream) {
  Eigen::Vector3i n = partition.getExtents() - 2 * partition.getGhostLayer();

  real* dfPtr = par->df->gpu_ptr(partition);
  real* df_tmpPtr = par->df_tmp->gpu_ptr(partition);
  real* dfTPtr = par->dfT->gpu_ptr(partition);
  real* dfT_tmpPtr = par->dfT_tmp->gpu_ptr(partition);

  Partition partitionNoGhostLayer(partition.getMin(), partition.getMax(),
                                  Eigen::Vector3i(0, 0, 0));
  real* avgSrcPtr = par->avg->gpu_ptr(partitionNoGhostLayer);
  real* avgDstPtr = par->avg_tmp->gpu_ptr(partitionNoGhostLayer);
  real* plotPtr = par->plot_tmp->gpu_ptr(partitionNoGhostLayer);
  voxel_t* voxelPtr = par->voxels->gpu_ptr(partitionNoGhostLayer);

  BoundaryCondition* bcsPtr = thrust::raw_pointer_cast(&(*par->bcs)[0]);

  dim3 gridSize(n.y(), n.z(), 1);
  dim3 blockSize(n.x(), 1, 1);
  ComputeAndPlotKernelInterior<<<gridSize, blockSize, 0, stream>>>(
      partition, dfPtr, df_tmpPtr, dfTPtr, dfT_tmpPtr, voxelPtr, bcsPtr,
      par->nu, par->C, par->nuT, par->Pr_t, par->gBetta, par->Tref, avgSrcPtr,
      avgDstPtr, displayQuantity, plotPtr);

  CUDA_CHECK_ERRORS("ComputeKernelInterior");
}

void KernelInterface::runComputeKernelBoundary(
    D3Q4::Enum direction,
    const Partition partition,
    ComputeParams* par,
    DisplayQuantity::Enum displayQuantity,
    hipStream_t stream) {
  Eigen::Vector3i n = partition.getExtents();

  real* dfPtr = par->df->gpu_ptr(partition);
  real* df_tmpPtr = par->df_tmp->gpu_ptr(partition);
  real* dfTPtr = par->dfT->gpu_ptr(partition);
  real* dfT_tmpPtr = par->dfT_tmp->gpu_ptr(partition);

  Partition partitionNoGhostLayer(partition.getMin(), partition.getMax(),
                                  Eigen::Vector3i(0, 0, 0));
  real* avgSrcPtr = par->avg->gpu_ptr(partitionNoGhostLayer);
  real* avgDstPtr = par->avg_tmp->gpu_ptr(partitionNoGhostLayer);
  real* plotPtr = par->plot_tmp->gpu_ptr(partitionNoGhostLayer);
  voxel_t* voxelPtr = par->voxels->gpu_ptr(partitionNoGhostLayer);

  BoundaryCondition* bcsPtr = thrust::raw_pointer_cast(&(*par->bcs)[0]);

  if (direction == D3Q4::X_AXIS) {
    dim3 gridSize(n.z(), 2, 1);
    dim3 blockSize(n.y(), 1, 1);
    ComputeAndPlotKernelBoundaryX<<<gridSize, blockSize, 0, stream>>>(
        partition, dfPtr, df_tmpPtr, dfTPtr, dfT_tmpPtr, voxelPtr, bcsPtr,
        par->nu, par->C, par->nuT, par->Pr_t, par->gBetta, par->Tref, avgSrcPtr,
        avgDstPtr, displayQuantity, plotPtr);
    CUDA_CHECK_ERRORS("ComputeKernelBoundaryX");
  }
  if (direction == D3Q4::Y_AXIS) {
    dim3 gridSize(n.z(), 2, 1);
    dim3 blockSize(n.x(), 1, 1);
    ComputeAndPlotKernelBoundaryY<<<gridSize, blockSize, 0, stream>>>(
        partition, dfPtr, df_tmpPtr, dfTPtr, dfT_tmpPtr, voxelPtr, bcsPtr,
        par->nu, par->C, par->nuT, par->Pr_t, par->gBetta, par->Tref, avgSrcPtr,
        avgDstPtr, displayQuantity, plotPtr);
    CUDA_CHECK_ERRORS("ComputeKernelBoundaryY");
  }
  if (direction == D3Q4::Z_AXIS) {
    dim3 gridSize(n.y(), 2, 1);
    dim3 blockSize(n.x(), 1, 1);
    ComputeAndPlotKernelBoundaryZ<<<gridSize, blockSize, 0, stream>>>(
        partition, dfPtr, df_tmpPtr, dfTPtr, dfT_tmpPtr, voxelPtr, bcsPtr,
        par->nu, par->C, par->nuT, par->Pr_t, par->gBetta, par->Tref, avgSrcPtr,
        avgDstPtr, displayQuantity, plotPtr);
    CUDA_CHECK_ERRORS("ComputeKernelBoundaryZ");
  }
}

std::vector<hipStream_t> KernelInterface::exchange(int srcDev,
                                                    Partition partition,
                                                    D3Q7::Enum direction) {
  ComputeParams* par = m_params.at(srcDev);
  Partition neighbour = par->df_tmp->getNeighbour(partition, direction);
  int dstDev = getPartitionDevice(neighbour);
  hipStream_t dfStream = getDfGhostLayerStream(srcDev, dstDev);
  hipStream_t dfTStream = getDfTGhostLayerStream(srcDev, dstDev);
  par->df_tmp->exchange(partition, m_params.at(dstDev)->df_tmp, neighbour,
                        direction, dfStream);
  par->dfT_tmp->exchange(partition, m_params.at(dstDev)->dfT_tmp, neighbour,
                         direction, dfTStream);
  CUDA_RT_CALL(hipStreamSynchronize(dfStream));
  CUDA_RT_CALL(hipStreamSynchronize(dfTStream));
  return std::vector<hipStream_t>{dfStream, dfTStream};
}

LatticeAverage KernelInterface::getAverage(VoxelVolume vol,
                                           uint64_t deltaTicks) {
  unsigned int offset = m_avgOffsets[vol];
  unsigned int size = vol.getNumVoxels();
  real temperature = m_avgs->getAverage(m_avgs->getPartition(), 0, offset, size,
                                        static_cast<real>(deltaTicks));
  real velocityX = m_avgs->getAverage(m_avgs->getPartition(), 1, offset, size,
                                      static_cast<real>(deltaTicks));
  real velocityY = m_avgs->getAverage(m_avgs->getPartition(), 2, offset, size,
                                      static_cast<real>(deltaTicks));
  real velocityZ = m_avgs->getAverage(m_avgs->getPartition(), 3, offset, size,
                                      static_cast<real>(deltaTicks));
  return LatticeAverage(temperature, velocityX, velocityY, velocityZ);
}

void KernelInterface::compute(DisplayQuantity::Enum displayQuantity,
                              Eigen::Vector3i slicePos,
                              real* sliceX,
                              real* sliceY,
                              real* sliceZ,
                              bool runSimulation) {
#pragma omp parallel num_threads(m_numDevices)
  {
    const int srcDev = omp_get_thread_num() % m_numDevices;

    CUDA_RT_CALL(hipSetDevice(srcDev));

    ComputeParams* par = m_params.at(srcDev);
    const Partition partition = getDevicePartition(srcDev);
    const Partition partitionNoGhostLayer(
        partition.getMin(), partition.getMax(), Eigen::Vector3i(0, 0, 0));

    const hipStream_t plotStream = getPlotStream(srcDev);
    const hipStream_t computeStream = getComputeStream(srcDev);
    const hipStream_t computeBoundaryStream = getComputeBoundaryStream(srcDev);
    const hipStream_t avgStream = getAvgStream(srcDev);

    // Compute LBM lattice boundary sites
    if (partition.getGhostLayer().x() > 0 && runSimulation) {
      runComputeKernelBoundary(D3Q4::X_AXIS, partition, par, displayQuantity,
                               computeBoundaryStream);
    }
    if (partition.getGhostLayer().y() > 0 && runSimulation) {
      runComputeKernelBoundary(D3Q4::Y_AXIS, partition, par, displayQuantity,
                               computeBoundaryStream);
    }
    if (partition.getGhostLayer().z() > 0 && runSimulation) {
      runComputeKernelBoundary(D3Q4::Z_AXIS, partition, par, displayQuantity,
                               computeBoundaryStream);
    }

    // Compute inner lattice sites (excluding boundaries)
    if (runSimulation)
      runComputeKernelInterior(partition, par, displayQuantity, computeStream);

    // Gather the plot to draw the display slices
    if (slicePos != Eigen::Vector3i(-1, -1, -1)) {
      par->plot->gatherSlice(slicePos, 0, 0, partitionNoGhostLayer, m_plot,
                             plotStream);
    }

    // Gather averages from GPU array
    if (runSimulation) {
      thrust::device_vector<real>* input =
          par->avg->getDeviceVector(partitionNoGhostLayer);
      thrust::device_vector<real>* output =
          m_avgs->getDeviceVector(m_avgs->getPartition());

      // thrust::gather_if(thrust::cuda::par.on(avgStream),
      // par->avgMap->begin(),
      //                   par->avgMap->end(), par->avgStencil->begin(),
      //                   input->begin(), output->begin());

      dim3 blockSize, gridSize;
      setExtents(par->avgMap->size(), BLOCK_SIZE_DEFAULT, &blockSize,
                 &gridSize);

      GatherKernel<<<gridSize, blockSize, 0, avgStream>>>(
          thrust::raw_pointer_cast(&(*par->avgMap)[0]), par->avgMap->size(),
          thrust::raw_pointer_cast(&(*par->avgStencil)[0]),
          thrust::raw_pointer_cast(&(*input)[0]),
          thrust::raw_pointer_cast(&(*output)[0]));
      CUDA_CHECK_ERRORS("GatherKernel");
      CUDA_RT_CALL(hipStreamSynchronize(avgStream));

      // TODO(gather_if fails when number of GPUs are 4-9 for some reason...)
      // for (int i = 0; i < par->avgMap->size(); i++) {
      //   int m = (*par->avgMap)[i];
      //   int s = (*par->avgStencil)[i];
      //   if (s) {
      //     real v = (*input)[m];
      //     (*output)[i] = v;
      //   }
      // }

      if (m_resetAvg) {
        CUDA_RT_CALL(hipStreamSynchronize(avgStream));
        par->avg->fill(0, avgStream);
        par->avg_tmp->fill(0, avgStream);
      }
    }

    // Wait for boundary lattice sites to finish computing
    CUDA_RT_CALL(hipStreamSynchronize(computeBoundaryStream));

    // Perform ghost layer exchanges
    if (partition.getGhostLayer().x() > 0 && runSimulation) {
      std::vector<hipStream_t> streamsPos =
          exchange(srcDev, partition, D3Q7::X_AXIS_POS);
      std::vector<hipStream_t> streamsNeg =
          exchange(srcDev, partition, D3Q7::X_AXIS_NEG);
      for (hipStream_t stream : streamsPos)
        CUDA_RT_CALL(hipStreamSynchronize(stream));
      for (hipStream_t stream : streamsNeg)
        CUDA_RT_CALL(hipStreamSynchronize(stream));
    }

#pragma omp barrier
    if (partition.getGhostLayer().y() > 0 && runSimulation) {
      std::vector<hipStream_t> streamsPos =
          exchange(srcDev, partition, D3Q7::Y_AXIS_POS);
      std::vector<hipStream_t> streamsNeg =
          exchange(srcDev, partition, D3Q7::Y_AXIS_NEG);
      for (hipStream_t stream : streamsPos)
        CUDA_RT_CALL(hipStreamSynchronize(stream));
      for (hipStream_t stream : streamsNeg)
        CUDA_RT_CALL(hipStreamSynchronize(stream));
    }

#pragma omp barrier
    if (partition.getGhostLayer().z() > 0 && runSimulation) {
      std::vector<hipStream_t> streamsPos =
          exchange(srcDev, partition, D3Q7::Z_AXIS_POS);
      std::vector<hipStream_t> streamsNeg =
          exchange(srcDev, partition, D3Q7::Z_AXIS_NEG);
      for (hipStream_t stream : streamsPos)
        CUDA_RT_CALL(hipStreamSynchronize(stream));
      for (hipStream_t stream : streamsNeg)
        CUDA_RT_CALL(hipStreamSynchronize(stream));
    }
    CUDA_RT_CALL(hipStreamSynchronize(plotStream));

#pragma omp barrier
    if (srcDev == 0 && slicePos != Eigen::Vector3i(-1, -1, -1)) {
      real* plot3dPtr = m_plot->gpu_ptr(m_plot->getPartition());
      dim3 blockSize, gridSize;

      setExtents(getExtents().y() * getExtents().z(), BLOCK_SIZE_DEFAULT,
                 &blockSize, &gridSize);
      SliceXRenderKernel<<<gridSize, blockSize, 0, plotStream>>>(
          plot3dPtr, getExtents().x(), getExtents().y(), getExtents().z(),
          sliceX, slicePos.x());
      CUDA_CHECK_ERRORS("SliceXRenderKernel");

      setExtents(getExtents().x() * getExtents().z(), BLOCK_SIZE_DEFAULT,
                 &blockSize, &gridSize);
      SliceYRenderKernel<<<gridSize, blockSize, 0, plotStream>>>(
          plot3dPtr, getExtents().x(), getExtents().y(), getExtents().z(),
          sliceY, slicePos.y());
      CUDA_CHECK_ERRORS("SliceYRenderKernel");

      setExtents(getExtents().x() * getExtents().y(), BLOCK_SIZE_DEFAULT,
                 &blockSize, &gridSize);
      SliceZRenderKernel<<<gridSize, blockSize, 0, plotStream>>>(
          plot3dPtr, getExtents().x(), getExtents().y(), getExtents().z(),
          sliceZ, slicePos.z());
      CUDA_CHECK_ERRORS("SliceZRenderKernel");
    }

    CUDA_RT_CALL(hipStreamSynchronize(computeStream));
    CUDA_RT_CALL(hipStreamSynchronize(avgStream));
    CUDA_RT_CALL(hipStreamSynchronize(plotStream));

#pragma omp barrier
    if (runSimulation) {
      DistributionFunction::swap(par->df, par->df_tmp);
      DistributionFunction::swap(par->dfT, par->dfT_tmp);
      DistributionFunction::swap(par->plot, par->plot_tmp);
      DistributionFunction::swap(par->avg, par->avg_tmp);
    }
  }
  m_resetAvg = false;
}

KernelInterface::KernelInterface(
    const int nx,
    const int ny,
    const int nz,
    const std::shared_ptr<ComputeParams> cmptParams,
    const std::shared_ptr<BoundaryConditions> bcs,
    const std::shared_ptr<VoxelArray> voxels,
    const std::shared_ptr<VoxelVolumeArray> avgVols,
    const int numDevices)
    : P2PLattice(nx, ny, nz, numDevices),
      m_params(numDevices),
      m_resetAvg(false) {
  std::cout << "Initializing LBM data structures..." << std::endl;
  CUDA_RT_CALL(hipSetDevice(0));
  CUDA_RT_CALL(hipFree(0));

  // Arrays for gathering distributed plot with back buffering
  m_plot = new DistributionArray<real>(1, nx, ny, nz);
  m_plot_tmp = new DistributionArray<real>(1, nx, ny, nz);
  m_plot->allocate();
  m_plot_tmp->allocate();
  m_plot->fill(0);
  m_plot_tmp->fill(0);

  // Array for gathering simulation averages
  int volumeVoxels = 0;
  for (int i = 0; i < avgVols->size(); i++) {
    VoxelVolume vol = avgVols->at(i);
    m_avgOffsets[vol] = volumeVoxels;
    Eigen::Vector3i ext = vol.getExtents();
    volumeVoxels += ext.x() * ext.y() * ext.z();
  }
  m_avgs = new DistributionArray<real>(4, volumeVoxels, 0, 0);
  m_avgs->allocate();
  m_avgs->fill(0);

  assert(m_avgs->size(m_avgs->getPartition()) == 4 * volumeVoxels);

  // Create maps and stencils for averaging with gather_if
  std::vector<int>* avgMaps[m_numDevices];
  std::vector<int>* avgStencils[m_numDevices];
  for (int srcDev = 0; srcDev < m_numDevices; srcDev++) {
    avgMaps[srcDev] = new std::vector<int>(4 * volumeVoxels, 0);
    avgStencils[srcDev] = new std::vector<int>(4 * volumeVoxels, 0);
  }
  int voxCount = 0;
  // Loop over all volumes
  for (int i = 0; i < avgVols->size(); i++) {
    VoxelVolume avg = avgVols->at(i);
    // Global minimum and maximum of volumes
    Eigen::Vector3i aMin = avg.getMin();
    Eigen::Vector3i aMax = avg.getMax();

    // Loop over all voxels in volume
    for (int z = aMin.z(); z < aMax.z(); z++)
      for (int y = aMin.y(); y < aMax.y(); y++)
        for (int x = aMin.x(); x < aMax.x(); x++) {
          // Voxel in volume in global coordinates
          Eigen::Vector3i vox = Eigen::Vector3i(x, y, z);
          // Loop over all lattice partitions
          for (int srcDev = 0; srcDev < m_numDevices; srcDev++) {
            const Partition latticePartition = getDevicePartition(srcDev);
            const Partition avgPartition(latticePartition.getMin(),
                                         latticePartition.getMax(),
                                         Eigen::Vector3i(0, 0, 0));

            const Eigen::Vector3i pMin = avgPartition.getMin();
            const Eigen::Vector3i pMax = avgPartition.getMax();
            const Eigen::Vector3i pExtents = avgPartition.getExtents();

            // Check if voxel is inside partition
            if ((pMin.x() <= vox.x() && vox.x() < pMax.x()) &&
                (pMin.y() <= vox.y() && vox.y() < pMax.y()) &&
                (pMin.z() <= vox.z() && vox.z() < pMax.z())) {
              // Convert voxel to local coordinate in partition
              Eigen::Vector3i srcPos = vox - pMin;
              // Loop over temperature (0) and each velocity (1-3)
              for (int q = 0; q < 4; q++) {
                // Convert local coordinate to array index
                int srcIndex = I4D(q, srcPos.x(), srcPos.y(), srcPos.z(),
                                   pExtents.x(), pExtents.y(), pExtents.z());
                int mapIdx = q * volumeVoxels + voxCount;
                avgMaps[srcDev]->at(mapIdx) = srcIndex;
                avgStencils[srcDev]->at(mapIdx) = 1;
                assert(srcIndex > 0 && srcIndex < avgPartition.getSize() * 4);
              }
              // Voxel can only be on one GPU...
              break;
            }
          }
          voxCount++;
        }
  }
  assert(voxCount == volumeVoxels);

  // Create one CPU thread per GPU
#pragma omp parallel num_threads(m_numDevices)
  {
    std::stringstream ss;

    const int srcDev = omp_get_thread_num();
    CUDA_RT_CALL(hipSetDevice(srcDev));
    CUDA_RT_CALL(hipFree(0));

    ComputeParams* par = new ComputeParams(*cmptParams);
    m_params.at(srcDev) = par;

    // Initialize distribution functions for temperature and velocity
    const Partition partition = getDevicePartition(srcDev);

    par->df = new DistributionFunction(19, nx, ny, nz, m_numDevices);
    par->df_tmp = new DistributionFunction(19, nx, ny, nz, m_numDevices);
    par->dfT = new DistributionFunction(7, nx, ny, nz, m_numDevices);
    par->dfT_tmp = new DistributionFunction(7, nx, ny, nz, m_numDevices);

    par->df->allocate(partition);
    par->df_tmp->allocate(partition);
    par->dfT->allocate(partition);
    par->dfT_tmp->allocate(partition);

    runInitKernel(par->df, par->dfT, partition, 1.0, 0, 0, 0, par->Tinit);
    runInitKernel(par->df_tmp, par->dfT_tmp, partition, 1.0, 0, 0, 0,
                  par->Tinit);
    ss << "Allocated partition " << partition << " on GPU" << srcDev
       << std::endl;

    // Arrays for averaging and plotting using back buffering
    const Partition partitionNoGhostLayer(
        partition.getMin(), partition.getMax(), Eigen::Vector3i(0, 0, 0));

    par->avg = new DistributionArray<real>(4, nx, ny, nz, m_numDevices);
    par->avg_tmp = new DistributionArray<real>(4, nx, ny, nz, m_numDevices);
    par->avg->allocate(partitionNoGhostLayer);
    par->avg_tmp->allocate(partitionNoGhostLayer);
    par->avg->fill(0);
    par->avg_tmp->fill(0);

    par->avgMap = new thrust::device_vector<int>(*avgMaps[srcDev]);
    par->avgStencil = new thrust::device_vector<int>(*avgStencils[srcDev]);

    // GPU local plot array with back buffering
    par->plot = new DistributionArray<real>(1, nx, ny, nz, m_numDevices);
    par->plot_tmp = new DistributionArray<real>(1, nx, ny, nz, m_numDevices);
    par->plot->allocate(partitionNoGhostLayer);
    par->plot_tmp->allocate(partitionNoGhostLayer);
    par->plot->fill(0);
    par->plot_tmp->fill(0);

    // Scatter voxel array into partitions
    par->voxels = new VoxelArray(nx, ny, nz, m_numDevices);
    par->voxels->allocate(partitionNoGhostLayer);
    par->voxels->scatter(*voxels, partitionNoGhostLayer);

    // Upload boundary conditions array
    par->bcs = new thrust::device_vector<BoundaryCondition>(*bcs);

    CUDA_RT_CALL(hipDeviceSynchronize());
    std::cout << ss.str();
  }  // end omp parallel

  std::cout << "LBM initialized" << std::endl;
}

void KernelInterface::uploadBCs(std::shared_ptr<BoundaryConditions> bcs) {
#pragma omp parallel num_threads(m_numDevices)
  {
    const int srcDev = omp_get_thread_num();
    CUDA_RT_CALL(hipSetDevice(srcDev));
    ComputeParams* par = m_params.at(srcDev);
    *par->bcs = *bcs;
  }
}

void KernelInterface::getMinMax(real* min,
                                real* max,
                                thrust::host_vector<real>* histogram) {
  *min = REAL_MAX;
  *max = REAL_MIN;
  thrust::host_vector<real> mins(m_numDevices);
  thrust::host_vector<real> maxes(m_numDevices);
  thrust::fill(histogram->begin(), histogram->end(), 0.0);

#pragma omp parallel num_threads(m_numDevices)
  {
    const int srcDev = omp_get_thread_num();
    CUDA_RT_CALL(hipSetDevice(srcDev));
    const Partition partition = getDevicePartition(srcDev);
    const Partition partitionNoGhostLayer(
        partition.getMin(), partition.getMax(), Eigen::Vector3i(0, 0, 0));
    ComputeParams* par = m_params.at(srcDev);
    mins[srcDev] = par->plot->getMin(partitionNoGhostLayer);
    maxes[srcDev] = par->plot->getMax(partitionNoGhostLayer);
#pragma omp barrier
#pragma omp single
    {
      *max = *thrust::max_element(maxes.begin(), maxes.end());
      *min = *thrust::min_element(mins.begin(), mins.end());
    }
    int nBins = histogram->size();
    thrust::host_vector<int> result(nBins);
    LatticeHistogram lHist;
    thrust::device_vector<real>* input =
        par->plot->getDeviceVector(partitionNoGhostLayer);
    lHist.calculate(input, *min, *max, nBins, &result);
#pragma omp critical
    for (int i = 0; i < nBins; i++) (*histogram)[i] += result[i];
  }
  for (int i = 0; i < histogram->size(); i++) (*histogram)[i] /= getSize();
}

void KernelInterface::resetDfs() {
#pragma omp parallel num_threads(m_numDevices)
  {
    const int srcDev = omp_get_thread_num();
    CUDA_RT_CALL(hipSetDevice(srcDev));
    const Partition partition = getDevicePartition(srcDev);
    ComputeParams* par = m_params.at(srcDev);
    runInitKernel(par->df, par->dfT, partition, 1.0, 0, 0, 0, par->Tinit);
    runInitKernel(par->df_tmp, par->dfT_tmp, partition, 1.0, 0, 0, 0,
                  par->Tinit);
  }
}
