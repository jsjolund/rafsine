#include "hip/hip_runtime.h"
#include "DistributionArray.hpp"

template <class T>
DistributionArray<T>::MemoryStore::MemoryStore(size_t size) {
  gpu = new thrust::device_vector<T>(size);
  cpu = new thrust::host_vector<T>(size);
}

template <class T>
void DistributionArray<T>::memcpy3DAsync(const DistributionArray<T>& src,
                                         Partition srcPart,
                                         unsigned int srcQ,
                                         vector3<unsigned int> srcPos,
                                         vector3<size_t> srcDim,
                                         DistributionArray<T>* dst,
                                         Partition dstPart,
                                         unsigned int dstQ,
                                         vector3<unsigned int> dstPos,
                                         vector3<size_t> dstDim,
                                         vector3<size_t> cpyExt,
                                         hipStream_t stream) {
  hipMemcpy3DParms cpy = {};
  // Source pointer
  cpy.srcPtr = make_hipPitchedPtr(
      src.gpu_ptr(srcPart, srcQ, srcPos.x(), srcPos.y(), srcPos.z()),
      srcDim.x() * sizeof(T), srcDim.x(), srcDim.y());
  // Destination pointer
  cpy.dstPtr = make_hipPitchedPtr(
      dst->gpu_ptr(dstPart, dstQ, dstPos.x(), dstPos.y(), dstPos.z()),
      dstDim.x() * sizeof(T), dstDim.x(), dstDim.y());
  // Extent of 3D copy
  cpy.extent = make_hipExtent(cpyExt.x() * sizeof(T), cpyExt.y(), cpyExt.z());
  cpy.kind = hipMemcpyDefault;

  CUDA_RT_CALL(hipMemcpy3DAsync(&cpy, stream));
}

template <class T>
DistributionArray<T>::DistributionArray(unsigned int q,
                                        unsigned int nx,
                                        unsigned int ny,
                                        unsigned int nz,
                                        unsigned int nd,
                                        unsigned int ghostLayerSize,
                                        D3Q4::Enum partitioning)
    : DistributedLattice(nx, ny, nz, nd, ghostLayerSize, partitioning),
      m_Q(q) {}

template <class T>
DistributionArray<T>::~DistributionArray() {
  for (std::pair<Partition, MemoryStore*> element : m_arrays) {
    if (element.second->gpu) delete element.second->gpu;
    if (element.second->cpu) delete element.second->cpu;
  }
}

template <class T>
void DistributionArray<T>::deallocate(MemoryType type, Partition partition) {
  if (partition.isEmpty()) partition = getPartition(0, 0, 0);
  if (m_arrays.find(partition) == m_arrays.end())
    throw std::out_of_range("Partition not allocated");
  MemoryStore* store = m_arrays[partition];
  if (type == DEVICE_MEMORY) {
    delete store->gpu;
    store->gpu = NULL;
  } else {
    delete store->cpu;
    store->cpu = NULL;
  }
}

template <class T>
void DistributionArray<T>::allocate(Partition partition) {
  if (partition.isEmpty()) partition = getPartition(0, 0, 0);
  if (m_arrays.find(partition) != m_arrays.end())
    throw std::out_of_range("Partition already allocated");
  int size = partition.getArrayStride() * m_Q;
  m_arrays[partition] = new MemoryStore(size);
}

template <class T>
std::vector<Partition> DistributionArray<T>::getAllocatedPartitions() {
  std::vector<Partition> partitions;
  for (std::pair<Partition, MemoryStore*> element : m_arrays)
    partitions.push_back(element.first);
  return partitions;
}

template <class T>
T DistributionArray<T>::getAverage(Partition partition,
                                   unsigned int q,
                                   unsigned int offset,
                                   unsigned int length,
                                   T divisor) {
  if (m_arrays.find(partition) == m_arrays.end())
    throw std::out_of_range("Partition not allocated");
  const int size = partition.getArrayStride();
  // download();
  // thrust::host_vector<T>* vec = m_arrays.at(partition)->cpu;
  // thrust::copy(vec->begin() + q * size, vec->begin() + (q + 1) * size,
  //              std::ostream_iterator<T>(std::cout, " "));
  // std::cout << std::endl;
  thrust::device_vector<T>* gpuVec = m_arrays.at(partition)->gpu;
  auto begin = gpuVec->begin() + q * size + offset;
  auto end = gpuVec->begin() + q * size + offset + length;
  return thrust::transform_reduce(
      begin, end, DistributionArray::division(static_cast<T>(length) * divisor),
      static_cast<T>(0), thrust::plus<T>());
}

// Fill the distribution function with a constant value for all nodes
template <class T>
void DistributionArray<T>::fill(T value, hipStream_t stream) {
  for (std::pair<Partition, MemoryStore*> element : m_arrays) {
    const int size = element.first.getArrayStride();
    thrust::device_vector<T>* gpuVec = element.second->gpu;
    thrust::fill(thrust::cuda::par.on(stream), gpuVec->begin(), gpuVec->end(),
                 value);
    // thrust::host_vector<T>* cpuVec = element.second->cpu;
    // thrust::fill(cpuVec->begin(), cpuVec->end(), value);
  }
}

template <class T>
void DistributionArray<T>::exchange(Partition partition,
                                    DistributionArray<T>* ndf,
                                    Partition neighbour,
                                    D3Q7::Enum direction,
                                    hipStream_t stream) {
  GhostLayerParameters segment = getGhostLayer(partition, neighbour, direction);

  for (int q : D3Q27ranks[direction]) {
    if (q >= getQ()) break;
    T* srcPtr = gpu_ptr(partition, q, segment.m_src.x(), segment.m_src.y(),
                        segment.m_src.z());
    T* dstPtr = ndf->gpu_ptr(neighbour, q, segment.m_dst.x(), segment.m_dst.y(),
                             segment.m_dst.z());
    CUDA_RT_CALL(hipMemcpy2DAsync(
        dstPtr, segment.m_dpitch, srcPtr, segment.m_spitch, segment.m_width,
        segment.m_height, hipMemcpyDefault, stream));
  }
}

// Read/write to specific allocated partition on CPU
template <class T>
T& DistributionArray<T>::operator()(Partition partition,
                                    unsigned int q,
                                    int x,
                                    int y,
                                    int z) {
  if (m_arrays.find(partition) == m_arrays.end())
    throw std::out_of_range("Partition not allocated");
  thrust::host_vector<T>* cpuVec = m_arrays.at(partition)->cpu;
  vector3<size_t> srcLatDim = partition.getArrayExtents();
  int idx = I4D(q, x, y, z, srcLatDim.x(), srcLatDim.y(), srcLatDim.z());
  return (*cpuVec)[idx];
}

// Read only, from specific allocated partition on CPU
template <class T>
T DistributionArray<T>::read(Partition partition,
                             unsigned int q,
                             int x,
                             int y,
                             int z) const {
  if (m_arrays.find(partition) == m_arrays.end())
    throw std::out_of_range("Partition not allocated");
  thrust::host_vector<T>* cpuVec = m_arrays.at(partition)->cpu;
  vector3<size_t> srcLatDim = partition.getArrayExtents();
  int idx = I4D(q, x, y, z, srcLatDim.x(), srcLatDim.y(), srcLatDim.z());
  return (*cpuVec)[idx];
}

template <class T>
T DistributionArray<T>::getMin(Partition partition) const {
  if (m_arrays.find(partition) == m_arrays.end())
    throw std::out_of_range("Partition not allocated");
  thrust::device_vector<T>* gpuVec = m_arrays.at(partition)->gpu;
  auto input_end =
      thrust::remove_if(gpuVec->begin(), gpuVec->end(), CUDA_isNaN());
  return *thrust::min_element(gpuVec->begin(), input_end);
}

template <class T>
T DistributionArray<T>::getMax(Partition partition) const {
  if (m_arrays.find(partition) == m_arrays.end())
    throw std::out_of_range("Partition not allocated");
  thrust::device_vector<T>* gpuVec = m_arrays.at(partition)->gpu;
  auto input_end =
      thrust::remove_if(gpuVec->begin(), gpuVec->end(), CUDA_isNaN());
  return *thrust::max_element(gpuVec->begin(), input_end);
}

// Return a pointer to the beginning of the GPU memory
template <class T>
T* DistributionArray<T>::gpu_ptr(Partition partition,
                                 unsigned int q,
                                 int x,
                                 int y,
                                 int z) const {
  if (m_arrays.find(partition) == m_arrays.end())
    throw std::out_of_range("Partition not allocated");
  thrust::device_vector<T>* gpuVec = m_arrays.at(partition)->gpu;
  vector3<size_t> srcLatDim = partition.getArrayExtents();
  size_t idx = I4D(q, x, y, z, srcLatDim.x(), srcLatDim.y(), srcLatDim.z());
  return thrust::raw_pointer_cast(&(*gpuVec)[idx]);
}

template <class T>
void DistributionArray<T>::scatter(const DistributionArray<T>& src,
                                   Partition dstPart,
                                   hipStream_t stream) {
  Partition srcPart = src.getPartition(0, 0, 0);

  vector3<size_t> dstLatDim = getExtents();
  vector3<size_t> srcLatDim = src.getExtents();
  vector3<size_t> srcDim = srcPart.getArrayExtents();

  // Lattices must have same size
  if (srcLatDim.x() != dstLatDim.x() || srcLatDim.y() != dstLatDim.y() ||
      srcLatDim.z() != dstLatDim.z() || getQ() != src.getQ())
    throw std::out_of_range("Lattice sizes must be equal");

  // The source partition must have the size of the entire lattice
  if (srcLatDim.x() != srcDim.x() || srcLatDim.y() != srcDim.y() ||
      srcLatDim.z() != srcDim.z())
    throw std::out_of_range(
        "Source sub lattice must have size of entire lattice");

  vector3<unsigned int> srcPos = dstPart.getMin();
  vector3<size_t> dstPos = dstPart.getGhostLayer();
  vector3<size_t> dstDim = dstPart.getArrayExtents();
  vector3<size_t> cpyExt = dstPart.getExtents();

  for (int q = 0; q < getQ(); q++) {
    memcpy3DAsync(src, srcPart, q, srcPos, srcDim, this, dstPart, q, dstPos,
                  dstDim, cpyExt, stream);
  }
}

template <class T>
void DistributionArray<T>::gather(Partition srcPart,
                                  DistributionArray<T>* dst,
                                  hipStream_t stream) {
  // Lattices must have same number of 3D arrays
  if (getQ() != dst->getQ())
    throw std::out_of_range("Lattice sizes must be equal");
  for (int q = 0; q < getQ(); q++) gather(q, q, srcPart, dst, stream);
}

template <class T>
void DistributionArray<T>::gather(unsigned int srcQ,
                                  unsigned int dstQ,
                                  Partition srcPart,
                                  DistributionArray<T>* dst,
                                  hipStream_t stream) {
  Partition dstPart = dst->getAllocatedPartitions().at(0);

  vector3<size_t> srcLatDim = getExtents();
  vector3<size_t> dstLatDim = dst->getExtents();
  vector3<size_t> dstDim = dstPart.getArrayExtents();
  // Lattices must have same size
  if (srcLatDim != dstLatDim)
    throw std::out_of_range("Lattice sizes must be equal");
  // The destination partition must have the size of the entire lattice
  if (srcLatDim != dstDim)
    throw std::out_of_range(
        "Destination sub lattice must have size of entire lattice");
  // Offset source position to exclude ghostLayers from copy
  vector3<size_t> srcPos = srcPart.getGhostLayer();
  // The destination is the global position of the source partition
  vector3<unsigned int> dstPos = srcPart.getMin();
  // Dimensions of source parition must include ghostLayers
  vector3<size_t> srcDim = srcPart.getArrayExtents();
  // Copy the full extent of the source partition, excluding ghostLayers
  vector3<size_t> cpyExt = srcPart.getExtents();
  memcpy3DAsync(*this, srcPart, srcQ, srcPos, srcDim, dst, dstPart, dstQ,
                dstPos, dstDim, cpyExt, stream);
}

template <class T>
void DistributionArray<T>::gather(vector3<unsigned int> globalMin,
                                  vector3<unsigned int> globalMax,
                                  unsigned int srcQ,
                                  unsigned int dstQ,
                                  Partition srcPart,
                                  DistributionArray<T>* dst,
                                  Partition dstPart,
                                  hipStream_t stream) {
  if (m_arrays.find(srcPart) == m_arrays.end())
    throw std::out_of_range("Partition not allocated");
  vector3<unsigned int> min, max;
  const int numVoxels = srcPart.intersect(globalMin, globalMax, &min, &max);
  // Size of the intersection
  const vector3<size_t> cpyExt = max - min;
  // Local position in partition
  const vector3<unsigned int> srcPos = min - srcPart.getMin();
  const vector3<size_t> srcDim = srcPart.getExtents();
  // Position in gather array
  const vector3<unsigned int> dstPos = srcPos + srcPart.getMin() - globalMin;
  const vector3<size_t> dstDim = globalMax - globalMin;
  if (numVoxels == 1) {
    // Read a single voxel
    T* srcGpuPtr = gpu_ptr(srcPart, srcQ, srcPos.x(), srcPos.y(), srcPos.z());
    T* dstGpuPtr =
        dst->gpu_ptr(dstPart, dstQ, dstPos.x(), dstPos.y(), dstPos.z());
    CUDA_RT_CALL(hipMemcpyAsync(dstGpuPtr, srcGpuPtr, sizeof(T),
                                 hipMemcpyDefault, stream));

  } else if (numVoxels > 1) {
    // Read a 3D volume
    memcpy3DAsync(*this, srcPart, srcQ, srcPos, srcDim, dst, dstPart, dstQ,
                  dstPos, dstDim, cpyExt, stream);
  }
}

template <class T>
void DistributionArray<T>::gatherSlice(vector3<unsigned int> slicePos,
                                       unsigned int srcQ,
                                       unsigned int dstQ,
                                       Partition srcPart,
                                       DistributionArray<T>* dst,
                                       hipStream_t stream) {
  vector3<unsigned int> offset = slicePos - srcPart.getMin();

  Partition dstPart = dst->getAllocatedPartitions().at(0);
  vector3<size_t> srcLatDim = getExtents();
  vector3<size_t> dstLatDim = dst->getExtents();
  vector3<size_t> dstDim = dstPart.getArrayExtents();

  // Lattices must have same size
  if (srcLatDim != dstLatDim)
    throw std::out_of_range("Lattice sizes must be equal");

  // The destination partition must have the size of the entire lattice
  if (srcLatDim != dstDim)
    throw std::out_of_range(
        "Destination sub lattice must have size of entire lattice");

  // Copy the three planes which intersect at slicePos
  if (slicePos.x() >= srcPart.getMin().x() &&
      slicePos.x() < srcPart.getMax().x()) {
    // Offset source position to exclude ghostLayers from copy
    vector3<unsigned int> srcPos = srcPart.getGhostLayer();
    srcPos.x() += offset.x();
    // The destination is the global position of the source partition
    vector3<unsigned int> dstPos = srcPart.getMin();
    dstPos.x() = slicePos.x();
    // Dimensions of source parition must include ghostLayers
    vector3<size_t> srcDim = srcPart.getArrayExtents();
    // Copy the full extent of the source partition, excluding ghostLayers
    vector3<size_t> cpyExt = srcPart.getExtents();
    cpyExt.x() = 1;
    memcpy3DAsync(*this, srcPart, srcQ, srcPos, srcDim, dst, dstPart, dstQ,
                  dstPos, dstDim, cpyExt, stream);
  }
  if (slicePos.y() >= srcPart.getMin().y() &&
      slicePos.y() < srcPart.getMax().y()) {
    vector3<unsigned int> srcPos = srcPart.getGhostLayer();
    srcPos.y() += offset.y();
    vector3<unsigned int> dstPos = srcPart.getMin();
    dstPos.y() = slicePos.y();
    vector3<size_t> srcDim = srcPart.getArrayExtents();
    vector3<size_t> cpyExt = srcPart.getExtents();
    cpyExt.y() = 1;
    memcpy3DAsync(*this, srcPart, srcQ, srcPos, srcDim, dst, dstPart, dstQ,
                  dstPos, dstDim, cpyExt, stream);
  }
  if (slicePos.z() >= srcPart.getMin().z() &&
      slicePos.z() < srcPart.getMax().z()) {
    vector3<unsigned int> srcPos = srcPart.getGhostLayer();
    srcPos.z() += offset.z();
    vector3<size_t> dstPos = srcPart.getMin();
    dstPos.z() = slicePos.z();
    vector3<size_t> srcDim = srcPart.getArrayExtents();
    vector3<size_t> cpyExt = srcPart.getExtents();
    cpyExt.z() = 1;
    memcpy3DAsync(*this, srcPart, srcQ, srcPos, srcDim, dst, dstPart, dstQ,
                  dstPos, dstDim, cpyExt, stream);
  }
}

// Upload the distributions functions from the CPU to the GPU
template <class T>
DistributionArray<T>& DistributionArray<T>::upload() {
  for (std::pair<Partition, MemoryStore*> element : m_arrays)
    *element.second->gpu = *element.second->cpu;
  return *this;
}

// Download the distributions functions from the GPU to the CPU
template <class T>
DistributionArray<T>& DistributionArray<T>::download() {
  for (std::pair<Partition, MemoryStore*> element : m_arrays)
    *element.second->cpu = *element.second->gpu;
  return *this;
}

template <class T>
DistributionArray<T>& DistributionArray<T>::operator=(
    const DistributionArray<T>& f) {
  if (getExtents() == f.getExtents()) {
    for (std::pair<Partition, MemoryStore*> element : m_arrays) {
      Partition partition = element.first;
      MemoryStore* v1 = element.second;
      if (f.m_arrays.find(partition) != f.m_arrays.end()) {
        MemoryStore* v2 = f.m_arrays.at(partition);
        // thrust::copy(v2.gpu->begin(), v2.gpu->end(), v1.gpu->begin());
        thrust::copy(v2->cpu->begin(), v2->cpu->end(), v1->cpu->begin());
      } else {
        throw std::out_of_range(
            "RHS must have allocated all partitions of LHS");
      }
    }
    return *this;
  }
  throw std::out_of_range("Distribution functions must have the same size");
}

// Static function to swap two DistributionArraysGroup
template <class T>
void DistributionArray<T>::swap(DistributionArray<T>* f1,
                                DistributionArray<T>* f2) {
  if (f1->m_arrays.size() == f2->m_arrays.size()) {
    for (std::pair<Partition, MemoryStore*> element : f1->m_arrays) {
      Partition partition = element.first;
      MemoryStore* v1 = element.second;
      if (f2->m_arrays.find(partition) != f2->m_arrays.end()) {
        MemoryStore* v2 = f2->m_arrays.at(partition);
        (*v1->gpu).swap(*v2->gpu);
        (*v1->cpu).swap(*v2->cpu);
      } else {
        throw std::out_of_range(
            "Cannot swap incompatible distribution functions");
      }
    }
    return;
  }
  throw std::out_of_range("Distribution functions must have the same size");
}
