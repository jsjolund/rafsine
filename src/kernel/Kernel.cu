#include "hip/hip_runtime.h"
#include "Kernel.hpp"

template <LBM::Enum method, D3Q4::Enum axis>
__global__ void ComputeKernel(const Partition partition,
                              real* __restrict__ df,
                              real* __restrict__ df_tmp,
                              real* __restrict__ dfT,
                              real* __restrict__ dfT_tmp,
                              real* __restrict__ dfTeff,
                              real* __restrict__ dfTeff_tmp,
                              const voxel_t* __restrict__ voxels,
                              BoundaryCondition* __restrict__ bcs,
                              const real dt,
                              const real nu,
                              const real C,
                              const real nuT,
                              const real Pr_t,
                              const real gBetta,
                              const real Tref,
                              real* __restrict__ averageSrc,
                              real* __restrict__ averageDst,
                              const DisplayQuantity::Enum displayQuantity,
                              real* __restrict__ plot) {
  const Eigen::Vector3i size = partition.getExtents();
  const Eigen::Vector3i gl = partition.getGhostLayer();

  // Compute node position from thread indexes
  int tx, ty, tz;

  switch (axis) {
    case D3Q4::X_AXIS:
      tx = blockIdx.y * (size.x() - 1);  // Might not be multiple of 32
      ty = threadIdx.x;
      tz = blockIdx.x;
      break;

    case D3Q4::Y_AXIS:
      tx = threadIdx.x;
      ty = blockIdx.y * (size.y() - 1);
      tz = blockIdx.x;
      break;

    case D3Q4::Z_AXIS:
      tx = threadIdx.x;
      ty = blockIdx.x;
      tz = blockIdx.y * (size.z() - 1);
      break;

    case D3Q4::ORIGIN:
      tx = threadIdx.x + gl.x();
      ty = blockIdx.x + gl.y();
      tz = blockIdx.y + gl.z();
      break;
  }

  // Check that the thread is inside the simulation domain
  if ((tx >= size.x()) || (ty >= size.y()) || (tz >= size.z())) return;

  Eigen::Vector3i pos(tx, ty, tz);
  const voxel_t voxelID = voxels[I3D(pos, size)];

  // Plot empty voxels
  if (voxelID == -1) {
    plot[I3D(pos, size)] = REAL_NAN;
    return;
  }

  const BoundaryCondition bc = bcs[voxelID];

  // Calculate array position for distribution functions (with ghostLayers)
  const int nx = size.x() + gl.x() * 2;
  const int ny = size.y() + gl.y() * 2;
  const int nz = size.z() + gl.z() * 2;

  const int x = pos.x() + gl.x();
  const int y = pos.y() + gl.y();
  const int z = pos.z() + gl.z();

  /// STEP 1 STREAMING
  // Store streamed distribution functions in registers
  // Modulo with wraparound for negative numbers
  const int xp = ((x + 1) % nx + nx) % nx;
  // x minus 1
  const int xm = ((x - 1) % nx + nx) % nx;
  // y plus 1
  const int yp = ((y + 1) % ny + ny) % ny;
  // y minus 1
  const int ym = ((y - 1) % ny + ny) % ny;
  // z plus 1
  const int zp = ((z + 1) % nz + nz) % nz;
  // z minus 1
  const int zm = ((z - 1) % nz + nz) % nz;

  real f0 = df3D(0, x, y, z, nx, ny, nz);
  real f1 = df3D(1, xm, y, z, nx, ny, nz);
  real f2 = df3D(2, xp, y, z, nx, ny, nz);
  real f3 = df3D(3, x, ym, z, nx, ny, nz);
  real f4 = df3D(4, x, yp, z, nx, ny, nz);
  real f5 = df3D(5, x, y, zm, nx, ny, nz);
  real f6 = df3D(6, x, y, zp, nx, ny, nz);
  real f7 = df3D(7, xm, ym, z, nx, ny, nz);
  real f8 = df3D(8, xp, yp, z, nx, ny, nz);
  real f9 = df3D(9, xm, yp, z, nx, ny, nz);
  real f10 = df3D(10, xp, ym, z, nx, ny, nz);
  real f11 = df3D(11, xm, y, zm, nx, ny, nz);
  real f12 = df3D(12, xp, y, zp, nx, ny, nz);
  real f13 = df3D(13, xm, y, zp, nx, ny, nz);
  real f14 = df3D(14, xp, y, zm, nx, ny, nz);
  real f15 = df3D(15, x, ym, zm, nx, ny, nz);
  real f16 = df3D(16, x, yp, zp, nx, ny, nz);
  real f17 = df3D(17, x, ym, zp, nx, ny, nz);
  real f18 = df3D(18, x, yp, zm, nx, ny, nz);

  real T0 = Tdf3D(0, x, y, z, nx, ny, nz);
  real T1 = Tdf3D(1, xm, y, z, nx, ny, nz);
  real T2 = Tdf3D(2, xp, y, z, nx, ny, nz);
  real T3 = Tdf3D(3, x, ym, z, nx, ny, nz);
  real T4 = Tdf3D(4, x, yp, z, nx, ny, nz);
  real T5 = Tdf3D(5, x, y, zm, nx, ny, nz);
  real T6 = Tdf3D(6, x, y, zp, nx, ny, nz);

  real* fs[19] = {&f0,  &f1,  &f2,  &f3,  &f4,  &f5,  &f6,  &f7,  &f8, &f9,
                  &f10, &f11, &f12, &f13, &f14, &f15, &f16, &f17, &f18};
  real* Ts[7] = {&T0, &T1, &T2, &T3, &T4, &T5, &T6};

  const real3 v =
      make_float3(bc.m_velocity.x(), bc.m_velocity.y(), bc.m_velocity.z());
  const real3 n =
      make_float3(bc.m_normal.x(), bc.m_normal.y(), bc.m_normal.z());

  if (bc.m_type == VoxelType::WALL) {
    // Half-way bounceback

// BC for velocity dfs
#pragma unroll
    for (int i = 1; i < 19; i++) {
      const real3 ei =
          make_float3(D3Q27[i * 3], D3Q27[i * 3 + 1], D3Q27[i * 3 + 2]);
      if (dot(ei, n) > 0.0) {
        *fs[i] = df3D(D3Q27Opposite[i], x, y, z, nx, ny, nz);
      }
    }
// BC for temperature dfs
#pragma unroll
    for (int i = 1; i < 7; i++) {
      const real3 ei =
          make_float3(D3Q27[i * 3], D3Q27[i * 3 + 1], D3Q27[i * 3 + 2]);
      if (dot(ei, n) > 0.0) {
        *Ts[i] = Tdf3D(D3Q27Opposite[i], x, y, z, nx, ny, nz);
      }
    }
    /////////////////////////////
  } else if (bc.m_type == VoxelType::INLET_CONSTANT ||
             bc.m_type == VoxelType::INLET_RELATIVE ||
             bc.m_type == VoxelType::INLET_ZERO_GRADIENT) {
// BC for velocity dfs
#pragma unroll
    for (int i = 1; i < 19; i++) {
      const real3 ei =
          make_float3(D3Q27[i * 3], D3Q27[i * 3 + 1], D3Q27[i * 3 + 2]);
      const real dot_vv = dot(v, v);
      if (dot(ei, n) > 0.0) {
        const real wi = D3Q19weights[i];
        const real rho_0 = 1.0;
        const real dot_eiv = dot(ei, v);
        // If the velocity is zero, use half-way bounceback instead
        if (length(v) == 0.0) {
          *fs[i] = df3D(D3Q27Opposite[i], x, y, z, nx, ny, nz);

        } else {
          *fs[i] =
              wi * rho_0 *
              (1.0 + 3.0 * dot_eiv + 4.5 * dot_eiv * dot_eiv - 1.5 * dot_vv);
        }
      }
    }
    // BC for temperature dfs
    if (bc.m_type == VoxelType::INLET_CONSTANT) {
#pragma unroll
      for (int i = 1; i < 7; i++) {
        const real3 ei =
            make_float3(D3Q27[i * 3], D3Q27[i * 3 + 1], D3Q27[i * 3 + 2]);
        const real wi = D3Q7weights[i];
        if (dot(ei, n) > 0.0) {
          *Ts[i] = wi * bc.m_temperature * (1.0 + 3.0 * dot(ei, v));
        }
      }
    } else if (bc.m_type == VoxelType::INLET_ZERO_GRADIENT) {
#pragma unroll
      for (int i = 1; i < 7; i++) {
        const real3 ei =
            make_float3(D3Q27[i * 3], D3Q27[i * 3 + 1], D3Q27[i * 3 + 2]);
        if (dot(ei, n) > 0.0) {
          // Approximate a first order expansion
          *Ts[i] = Tdf3D(i, x + bc.m_normal.x(), y + bc.m_normal.y(),
                         z + bc.m_normal.z(), nx, ny, nz);
        }
      }
    } else if (bc.m_type == VoxelType::INLET_RELATIVE) {
      // Compute macroscopic temperature at the relative position
      real Tamb = 0;
#pragma unroll
      for (int i = 1; i < 7; i++) {
        Tamb += Tdf3D(i, x + bc.m_rel_pos.x(), y + bc.m_rel_pos.y(),
                      z + bc.m_rel_pos.z(), nx, ny, nz);
      }
      // Internal temperature
      real Teff_old = dfTeff[I4D(0, x, y, z, nx, ny, nz)];

      real Teff_new = bc.m_tau1 / (bc.m_tau1 + dt) * Teff_old +
                      dt / (bc.m_tau1 + dt) *
                          (Tamb + (1.0 - bc.m_lambda) * bc.m_temperature);
      real Tnew =
          Tamb + bc.m_temperature +
          bc.m_tau2 / (bc.m_tau1 + dt) *
              (Teff_old - Tamb - (1.0 - bc.m_lambda) * bc.m_temperature);

#pragma unroll
      for (int i = 1; i < 7; i++) {
        const real3 ei =
            make_float3(D3Q27[i * 3], D3Q27[i * 3 + 1], D3Q27[i * 3 + 2]);
        const real wi = D3Q7weights[i];

        if (dot(ei, n) > 0.0) { *Ts[i] = Tnew * wi * (1.0 + 3.0 * dot(ei, v)); }
      }
      dfTeff_tmp[I4D(0, x, y, z, nx, ny, nz)] = Teff_new;
    }
  }

  PhysicalQuantity phy = {.rho = 0, .T = 0, .vx = 0, .vy = 0, .vz = 0};

  switch (method) {
    case LBM::BGK:
      computeBGK(x, y, z, nx, ny, nz, nu, nuT, C, Pr_t, gBetta, Tref, f0, f1,
                 f2, f3, f4, f5, f6, f7, f8, f9, f10, f11, f12, f13, f14, f15,
                 f16, f17, f18, T0, T1, T2, T3, T4, T5, T6, df_tmp, dfT_tmp,
                 &phy);
      break;

    case LBM::MRT:
      computeMRT(x, y, z, nx, ny, nz, nu, nuT, C, Pr_t, gBetta, Tref, f0, f1,
                 f2, f3, f4, f5, f6, f7, f8, f9, f10, f11, f12, f13, f14, f15,
                 f16, f17, f18, T0, T1, T2, T3, T4, T5, T6, df_tmp, dfT_tmp,
                 &phy);
      break;
  }

  // Average temperature and velocity
  averageDst[I4D(0, pos, size)] = averageSrc[I4D(0, pos, size)] + phy.T;
  averageDst[I4D(1, pos, size)] = averageSrc[I4D(1, pos, size)] + phy.vx;
  averageDst[I4D(2, pos, size)] = averageSrc[I4D(2, pos, size)] + phy.vy;
  averageDst[I4D(3, pos, size)] = averageSrc[I4D(3, pos, size)] + phy.vz;

  switch (displayQuantity) {
    case DisplayQuantity::VELOCITY_NORM:
      plot[I3D(pos, size)] =
          sqrt(phy.vx * phy.vx + phy.vy * phy.vy + phy.vz * phy.vz);
      break;
    case DisplayQuantity::DENSITY:
      plot[I3D(pos, size)] = phy.rho;
      break;
    case DisplayQuantity::TEMPERATURE:
      plot[I3D(pos, size)] = phy.T;
      break;
  }
}

#define LBM_CONFIGS         \
  X(LBM::BGK, D3Q4::ORIGIN) \
  X(LBM::BGK, D3Q4::X_AXIS) \
  X(LBM::BGK, D3Q4::Y_AXIS) \
  X(LBM::BGK, D3Q4::Z_AXIS) \
  X(LBM::MRT, D3Q4::ORIGIN) \
  X(LBM::MRT, D3Q4::X_AXIS) \
  X(LBM::MRT, D3Q4::Y_AXIS) \
  X(LBM::MRT, D3Q4::Z_AXIS)

#define X(METHOD, AXIS)                                                  \
  template __global__ void ComputeKernel<METHOD, AXIS>(                  \
      const Partition partition, real* __restrict__ df,                  \
      real* __restrict__ df_tmp, real* __restrict__ dfT,                 \
      real* __restrict__ dfT_tmp, real* __restrict__ dfTeff,             \
      real* __restrict__ dfTeff_tmp, const voxel_t* __restrict__ voxels, \
      BoundaryCondition* __restrict__ bcs, const real dt, const real nu, \
      const real C, const real nuT, const real Pr_t, const real gBetta,  \
      const real Tref, real* __restrict__ averageSrc,                    \
      real* __restrict__ averageDst,                                     \
      const DisplayQuantity::Enum displayQuantity, real* __restrict__ plot);
LBM_CONFIGS
#undef X
