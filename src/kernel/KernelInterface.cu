#include "hip/hip_runtime.h"
#include "KernelInterface.hpp"

void KernelInterface::runInitKernel(DistributionFunction* df,
                                    DistributionFunction* dfT,
                                    Partition partition,
                                    float rho,
                                    float vx,
                                    float vy,
                                    float vz,
                                    float T) {
  float sq_term = -1.5f * (vx * vx + vy * vy + vz * vz);
  vector3<int> n = partition.getArrayExtents();
  dim3 gridSize(n.y(), n.z(), 1);
  dim3 blockSize(n.x(), 1, 1);
  real* dfPtr = df->gpu_ptr(partition);
  real* dfTPtr = dfT->gpu_ptr(partition);

  InitKernel<<<gridSize, blockSize>>>(dfPtr, dfTPtr, n.x(), n.y(), n.z(), rho,
                                      vx, vy, vz, T, sq_term);
  CUDA_CHECK_ERRORS("InitKernel");
}

void KernelInterface::runComputeKernelInterior(
    const Partition partition,
    SimulationParams* param,
    SimulationState* state,
    DisplayQuantity::Enum displayQuantity,
    hipStream_t stream) {
  vector3<int> n = partition.getExtents() - partition.getGhostLayer() * 2;

  real* dfPtr = state->df->gpu_ptr(partition);
  real* df_tmpPtr = state->df_tmp->gpu_ptr(partition);
  real* dfTPtr = state->dfT->gpu_ptr(partition);
  real* dfT_tmpPtr = state->dfT_tmp->gpu_ptr(partition);
  real* dfTeffPtr = state->dfTeff->gpu_ptr(partition);
  real* dfTeff_tmpPtr = state->dfTeff_tmp->gpu_ptr(partition);

  Partition partitionNoGhostLayer(partition.getMin(), partition.getMax(),
                                  vector3<int>(0, 0, 0));
  real* avgSrcPtr = state->avg->gpu_ptr(partitionNoGhostLayer);
  real* avgDstPtr = state->avg_tmp->gpu_ptr(partitionNoGhostLayer);
  real* plotPtr = state->plot_tmp->gpu_ptr(partitionNoGhostLayer);
  voxel_t* voxelPtr = state->voxels->gpu_ptr(partitionNoGhostLayer);

  BoundaryCondition* bcsPtr = thrust::raw_pointer_cast(&(*state->bcs)[0]);

  dim3 gridSize(n.y(), n.z(), 1);
  dim3 blockSize(n.x(), 1, 1);
  if (m_method == LBM::BGK)
    ComputeKernel<LBM::BGK, D3Q4::ORIGIN><<<gridSize, blockSize, 0, stream>>>(
        partition, dfPtr, df_tmpPtr, dfTPtr, dfT_tmpPtr, dfTeffPtr,
        dfTeff_tmpPtr, voxelPtr, bcsPtr, m_dt, param->nu, param->C, param->nuT,
        param->Pr_t, param->gBetta, param->Tref, avgSrcPtr, avgDstPtr,
        displayQuantity, plotPtr);
  else if (m_method == LBM::MRT)
    ComputeKernel<LBM::MRT, D3Q4::ORIGIN><<<gridSize, blockSize, 0, stream>>>(
        partition, dfPtr, df_tmpPtr, dfTPtr, dfT_tmpPtr, dfTeffPtr,
        dfTeff_tmpPtr, voxelPtr, bcsPtr, m_dt, param->nu, param->C, param->nuT,
        param->Pr_t, param->gBetta, param->Tref, avgSrcPtr, avgDstPtr,
        displayQuantity, plotPtr);
  CUDA_CHECK_ERRORS("ComputeKernelInterior");
}

void KernelInterface::runComputeKernelBoundary(
    D3Q4::Enum direction,
    const Partition partition,
    SimulationParams* param,
    SimulationState* state,
    DisplayQuantity::Enum displayQuantity,
    hipStream_t stream) {
  vector3<int> n = partition.getExtents();

  real* dfPtr = state->df->gpu_ptr(partition);
  real* df_tmpPtr = state->df_tmp->gpu_ptr(partition);
  real* dfTPtr = state->dfT->gpu_ptr(partition);
  real* dfT_tmpPtr = state->dfT_tmp->gpu_ptr(partition);
  real* dfTeffPtr = state->dfTeff->gpu_ptr(partition);
  real* dfTeff_tmpPtr = state->dfTeff_tmp->gpu_ptr(partition);

  Partition partitionNoGhostLayer(partition.getMin(), partition.getMax(),
                                  vector3<int>(0, 0, 0));
  real* avgSrcPtr = state->avg->gpu_ptr(partitionNoGhostLayer);
  real* avgDstPtr = state->avg_tmp->gpu_ptr(partitionNoGhostLayer);
  real* plotPtr = state->plot_tmp->gpu_ptr(partitionNoGhostLayer);
  voxel_t* voxelPtr = state->voxels->gpu_ptr(partitionNoGhostLayer);

  BoundaryCondition* bcsPtr = thrust::raw_pointer_cast(&(*state->bcs)[0]);

  if (direction == D3Q4::X_AXIS) {
    dim3 gridSize(n.z(), 2, 1);
    dim3 blockSize(n.y(), 1, 1);
    if (m_method == LBM::BGK)
      ComputeKernel<LBM::BGK, D3Q4::X_AXIS><<<gridSize, blockSize, 0, stream>>>(
          partition, dfPtr, df_tmpPtr, dfTPtr, dfT_tmpPtr, dfTeffPtr,
          dfTeff_tmpPtr, voxelPtr, bcsPtr, m_dt, param->nu, param->C,
          param->nuT, param->Pr_t, param->gBetta, param->Tref, avgSrcPtr,
          avgDstPtr, displayQuantity, plotPtr);
    else if (m_method == LBM::MRT)
      ComputeKernel<LBM::MRT, D3Q4::X_AXIS><<<gridSize, blockSize, 0, stream>>>(
          partition, dfPtr, df_tmpPtr, dfTPtr, dfT_tmpPtr, dfTeffPtr,
          dfTeff_tmpPtr, voxelPtr, bcsPtr, m_dt, param->nu, param->C,
          param->nuT, param->Pr_t, param->gBetta, param->Tref, avgSrcPtr,
          avgDstPtr, displayQuantity, plotPtr);
    CUDA_CHECK_ERRORS("ComputeKernelBoundaryX");
  }
  if (direction == D3Q4::Y_AXIS) {
    dim3 gridSize(n.z(), 2, 1);
    dim3 blockSize(n.x(), 1, 1);
    if (m_method == LBM::BGK)
      ComputeKernel<LBM::BGK, D3Q4::Y_AXIS><<<gridSize, blockSize, 0, stream>>>(
          partition, dfPtr, df_tmpPtr, dfTPtr, dfT_tmpPtr, dfTeffPtr,
          dfTeff_tmpPtr, voxelPtr, bcsPtr, m_dt, param->nu, param->C,
          param->nuT, param->Pr_t, param->gBetta, param->Tref, avgSrcPtr,
          avgDstPtr, displayQuantity, plotPtr);
    else if (m_method == LBM::MRT)
      ComputeKernel<LBM::MRT, D3Q4::Y_AXIS><<<gridSize, blockSize, 0, stream>>>(
          partition, dfPtr, df_tmpPtr, dfTPtr, dfT_tmpPtr, dfTeffPtr,
          dfTeff_tmpPtr, voxelPtr, bcsPtr, m_dt, param->nu, param->C,
          param->nuT, param->Pr_t, param->gBetta, param->Tref, avgSrcPtr,
          avgDstPtr, displayQuantity, plotPtr);
    CUDA_CHECK_ERRORS("ComputeKernelBoundaryY");
  }
  if (direction == D3Q4::Z_AXIS) {
    dim3 gridSize(n.y(), 2, 1);
    dim3 blockSize(n.x(), 1, 1);
    if (m_method == LBM::BGK)
      ComputeKernel<LBM::BGK, D3Q4::Z_AXIS><<<gridSize, blockSize, 0, stream>>>(
          partition, dfPtr, df_tmpPtr, dfTPtr, dfT_tmpPtr, dfTeffPtr,
          dfTeff_tmpPtr, voxelPtr, bcsPtr, m_dt, param->nu, param->C,
          param->nuT, param->Pr_t, param->gBetta, param->Tref, avgSrcPtr,
          avgDstPtr, displayQuantity, plotPtr);
    else if (m_method == LBM::MRT)
      ComputeKernel<LBM::MRT, D3Q4::Z_AXIS><<<gridSize, blockSize, 0, stream>>>(
          partition, dfPtr, df_tmpPtr, dfTPtr, dfT_tmpPtr, dfTeffPtr,
          dfTeff_tmpPtr, voxelPtr, bcsPtr, m_dt, param->nu, param->C,
          param->nuT, param->Pr_t, param->gBetta, param->Tref, avgSrcPtr,
          avgDstPtr, displayQuantity, plotPtr);
    CUDA_CHECK_ERRORS("ComputeKernelBoundaryZ");
  }
}

std::vector<hipStream_t> KernelInterface::exchange(int srcDev,
                                                    Partition partition,
                                                    D3Q7::Enum direction) {
  SimulationState* state = m_state.at(srcDev);
  Partition neighbour = state->df_tmp->getNeighbour(partition, direction);
  int dstDev = getPartitionDevice(neighbour);
  hipStream_t dfStream = getDfGhostLayerStream(srcDev, dstDev);
  hipStream_t dfTStream = getDfTGhostLayerStream(srcDev, dstDev);
  state->df_tmp->exchange(partition, m_state.at(dstDev)->df_tmp, neighbour,
                          direction, dfStream);
  state->dfT_tmp->exchange(partition, m_state.at(dstDev)->dfT_tmp, neighbour,
                           direction, dfTStream);
  state->dfTeff_tmp->exchange(partition, m_state.at(dstDev)->dfTeff_tmp,
                              neighbour, direction, dfTStream);
  CUDA_RT_CALL(hipStreamSynchronize(dfStream));
  CUDA_RT_CALL(hipStreamSynchronize(dfTStream));
  return std::vector<hipStream_t>{dfStream, dfTStream};
}

void KernelInterface::calculateAverages() {
  thrust::host_vector<real>* avgs =
      m_avgs->getHostVector(m_avgs->getPartition());
  for (int srcDev = 0; srcDev < m_nd; srcDev++) {
    SimulationState* state = m_state.at(srcDev);
    thrust::host_vector<real> avgPartial =
        *state->avgResult->getHostVector(state->avgResult->getPartition());
    thrust::host_vector<int> avgStencil = *state->avgStencil;
    thrust::counting_iterator<int> iter(0);

    thrust::gather_if(iter, iter + avgPartial.size(), avgStencil.begin(),
                      avgPartial.begin(), avgs->begin());
  }
  m_avgs->upload();
}

LatticeAverage KernelInterface::getAverage(VoxelVolume vol,
                                           uint64_t deltaTicks) {
  unsigned int offset = m_avgOffsets[vol];
  unsigned int size = vol.getNumVoxels();
  real ticks = static_cast<real>(deltaTicks);
  Partition partition = m_avgs->getPartition();
  real temperature = m_avgs->getAverage(partition, 0, offset, size, ticks);
  real velocityX = m_avgs->getAverage(partition, 1, offset, size, ticks);
  real velocityY = m_avgs->getAverage(partition, 2, offset, size, ticks);
  real velocityZ = m_avgs->getAverage(partition, 3, offset, size, ticks);
  return LatticeAverage(temperature, velocityX, velocityY, velocityZ);
}

void KernelInterface::compute(DisplayQuantity::Enum displayQuantity,
                              vector3<int> slicePos,
                              real* sliceX,
                              real* sliceY,
                              real* sliceZ,
                              bool runSimulation) {
#pragma omp parallel num_threads(m_nd)
  {
    const int srcDev = omp_get_thread_num() % m_nd;

    CUDA_RT_CALL(hipSetDevice(srcDev));

    SimulationParams* param = m_params.at(srcDev);
    SimulationState* state = m_state.at(srcDev);
    const Partition partition = getDevicePartition(srcDev);
    const Partition partitionNoGhostLayer(
        partition.getMin(), partition.getMax(), vector3<int>(0, 0, 0));

    const hipStream_t plotStream = getPlotStream(srcDev);
    const hipStream_t computeStream = getComputeStream(srcDev);
    const hipStream_t computeBoundaryStream = getComputeBoundaryStream(srcDev);
    const hipStream_t avgStream = getAvgStream(srcDev);

    // Compute LBM lattice boundary sites
    if (partition.getGhostLayer().x() > 0 && runSimulation) {
      runComputeKernelBoundary(D3Q4::X_AXIS, partition, param, state,
                               displayQuantity, computeBoundaryStream);
    }
    if (partition.getGhostLayer().y() > 0 && runSimulation) {
      runComputeKernelBoundary(D3Q4::Y_AXIS, partition, param, state,
                               displayQuantity, computeBoundaryStream);
    }
    if (partition.getGhostLayer().z() > 0 && runSimulation) {
      runComputeKernelBoundary(D3Q4::Z_AXIS, partition, param, state,
                               displayQuantity, computeBoundaryStream);
    }

    // Compute inner lattice sites (excluding boundaries)
    if (runSimulation)
      runComputeKernelInterior(partition, param, state, displayQuantity,
                               computeStream);

    // Gather the plot to draw the display slices
    if (slicePos != vector3<int>(-1, -1, -1)) {
      state->plot->gatherSlice(slicePos, 0, 0, partitionNoGhostLayer, m_plot,
                               plotStream);
    }

    // Gather averages from GPU array
    if (runSimulation) {
      thrust::device_vector<real>* input =
          state->avg->getDeviceVector(partitionNoGhostLayer);
      thrust::device_vector<real>* output =
          state->avgResult->getDeviceVector(state->avgResult->getPartition());
      thrust::gather(thrust::cuda::par.on(avgStream), state->avgMap->begin(),
                     state->avgMap->end(), input->begin(), output->begin());
      state->avgResult->download();
      if (m_resetAvg) {
        state->avg->fill(0, avgStream);
        state->avg_tmp->fill(0, avgStream);
      }
    }

    // Wait for boundary lattice sites to finish computing
    CUDA_RT_CALL(hipStreamSynchronize(computeBoundaryStream));

    // Perform ghost layer exchanges
    if (partition.getGhostLayer().x() > 0 && runSimulation) {
      std::vector<hipStream_t> streamsPos =
          exchange(srcDev, partition, D3Q7::X_AXIS_POS);
      std::vector<hipStream_t> streamsNeg =
          exchange(srcDev, partition, D3Q7::X_AXIS_NEG);
      for (hipStream_t stream : streamsPos)
        CUDA_RT_CALL(hipStreamSynchronize(stream));
      for (hipStream_t stream : streamsNeg)
        CUDA_RT_CALL(hipStreamSynchronize(stream));
    }

#pragma omp barrier
    if (partition.getGhostLayer().y() > 0 && runSimulation) {
      std::vector<hipStream_t> streamsPos =
          exchange(srcDev, partition, D3Q7::Y_AXIS_POS);
      std::vector<hipStream_t> streamsNeg =
          exchange(srcDev, partition, D3Q7::Y_AXIS_NEG);
      for (hipStream_t stream : streamsPos)
        CUDA_RT_CALL(hipStreamSynchronize(stream));
      for (hipStream_t stream : streamsNeg)
        CUDA_RT_CALL(hipStreamSynchronize(stream));
    }

#pragma omp barrier
    if (partition.getGhostLayer().z() > 0 && runSimulation) {
      std::vector<hipStream_t> streamsPos =
          exchange(srcDev, partition, D3Q7::Z_AXIS_POS);
      std::vector<hipStream_t> streamsNeg =
          exchange(srcDev, partition, D3Q7::Z_AXIS_NEG);
      for (hipStream_t stream : streamsPos)
        CUDA_RT_CALL(hipStreamSynchronize(stream));
      for (hipStream_t stream : streamsNeg)
        CUDA_RT_CALL(hipStreamSynchronize(stream));
    }
    CUDA_RT_CALL(hipStreamSynchronize(plotStream));

#pragma omp barrier
    if (srcDev == 0 && slicePos != vector3<int>(-1, -1, -1)) {
      real* plot3dPtr = m_plot->gpu_ptr(m_plot->getPartition());
      dim3 blockSize, gridSize;

      vector3<int> n = getExtents();
      setExtents(n.y() * n.z(), BLOCK_SIZE_DEFAULT, &blockSize, &gridSize);
      SliceXRenderKernel<<<gridSize, blockSize, 0, plotStream>>>(
          plot3dPtr, n.x(), n.y(), n.z(), sliceX, slicePos.x());
      CUDA_CHECK_ERRORS("SliceXRenderKernel");

      setExtents(n.x() * n.z(), BLOCK_SIZE_DEFAULT, &blockSize, &gridSize);
      SliceYRenderKernel<<<gridSize, blockSize, 0, plotStream>>>(
          plot3dPtr, n.x(), n.y(), n.z(), sliceY, slicePos.y());
      CUDA_CHECK_ERRORS("SliceYRenderKernel");

      setExtents(n.x() * n.y(), BLOCK_SIZE_DEFAULT, &blockSize, &gridSize);
      SliceZRenderKernel<<<gridSize, blockSize, 0, plotStream>>>(
          plot3dPtr, n.x(), n.y(), n.z(), sliceZ, slicePos.z());
      CUDA_CHECK_ERRORS("SliceZRenderKernel");
    }

    CUDA_RT_CALL(hipStreamSynchronize(computeStream));
    CUDA_RT_CALL(hipStreamSynchronize(avgStream));
    CUDA_RT_CALL(hipStreamSynchronize(plotStream));

#pragma omp barrier
    if (runSimulation) {
      DistributionFunction::swap(state->df, state->df_tmp);
      DistributionFunction::swap(state->dfT, state->dfT_tmp);
      DistributionFunction::swap(state->dfTeff, state->dfTeff_tmp);
      DistributionFunction::swap(state->plot, state->plot_tmp);
      DistributionFunction::swap(state->avg, state->avg_tmp);
    }
  }
  m_resetAvg = false;
}

KernelInterface::KernelInterface(
    const int nx,
    const int ny,
    const int nz,
    const real dt,
    const std::shared_ptr<SimulationParams> cmptParams,
    const std::shared_ptr<BoundaryConditions> bcs,
    const std::shared_ptr<VoxelArray> voxels,
    const std::shared_ptr<VoxelVolumeArray> avgVols,
    const int nd,
    const LBM::Enum method,
    const D3Q4::Enum partitioning)
    : P2PLattice(nx, ny, nz, nd, partitioning),
      m_params(nd),
      m_state(nd),
      m_method(method),
      m_resetAvg(false),
      m_dt(dt) {
  std::cout << "Initializing LBM data structures..." << std::endl;
  CUDA_RT_CALL(hipSetDevice(0));
  CUDA_RT_CALL(hipFree(0));

  // Arrays for gathering distributed plot with back buffering
  m_plot = new DistributionArray<real>(1, nx, ny, nz, 1, 0, partitioning);
  m_plot_tmp = new DistributionArray<real>(1, nx, ny, nz, 1, 0, partitioning);
  m_plot->allocate();
  m_plot_tmp->allocate();
  m_plot->fill(0);
  m_plot_tmp->fill(0);

  // Array for gathering simulation averages
  int numAvgVoxels = 0;
  for (int i = 0; i < avgVols->size(); i++) {
    VoxelVolume vol = avgVols->at(i);
    m_avgOffsets[vol] = numAvgVoxels;
    vector3<int> ext = vol.getExtents();
    numAvgVoxels += ext.x() * ext.y() * ext.z();
  }
  m_avgs =
      new DistributionArray<real>(4, numAvgVoxels, 0, 0, 1, 0, partitioning);
  m_avgs->allocate();
  m_avgs->fill(0);

  // Create maps and stencils for averaging with gather_if
  std::vector<int>* avgMaps[nd];
  std::vector<int>* avgStencils[nd];
  for (int srcDev = 0; srcDev < nd; srcDev++) {
    avgMaps[srcDev] = new std::vector<int>(4 * numAvgVoxels, 0);
    avgStencils[srcDev] = new std::vector<int>(4 * numAvgVoxels, 0);
  }
  int voxCounter = 0;
  // Loop over all volumes
  for (int i = 0; i < avgVols->size(); i++) {
    VoxelVolume avg = avgVols->at(i);
    // Global minimum and maximum of volumes
    vector3<int> aMin = avg.getMin();
    vector3<int> aMax = avg.getMax();

    // Loop over all voxels in volume
    for (int z = aMin.z(); z < aMax.z(); z++)
      for (int y = aMin.y(); y < aMax.y(); y++)
        for (int x = aMin.x(); x < aMax.x(); x++) {
          // Voxel in volume in global coordinates
          vector3<int> vox = vector3<int>(x, y, z);
          // Loop over all lattice partitions
          for (int srcDev = 0; srcDev < nd; srcDev++) {
            const Partition latticePartition = getDevicePartition(srcDev);
            const Partition avgPartition(latticePartition.getMin(),
                                         latticePartition.getMax(),
                                         vector3<int>(0, 0, 0));

            const vector3<int> pMin = avgPartition.getMin();
            const vector3<int> pMax = avgPartition.getMax();
            const vector3<int> pExt = avgPartition.getExtents();

            // Check if voxel is inside partition
            if ((pMin.x() <= vox.x() && vox.x() < pMax.x()) &&
                (pMin.y() <= vox.y() && vox.y() < pMax.y()) &&
                (pMin.z() <= vox.z() && vox.z() < pMax.z())) {
              // Convert voxel to local coordinate in partition
              vector3<int> srcPos = vox - pMin;
              // Loop over temperature (0) and each velocity (1-3)
              for (int q = 0; q < 4; q++) {
                // Convert local coordinate to array index
                int srcIndex = I4D(q, srcPos.x(), srcPos.y(), srcPos.z(),
                                   pExt.x(), pExt.y(), pExt.z());
                int mapIdx = q * numAvgVoxels + voxCounter;
                avgMaps[srcDev]->at(mapIdx) = srcIndex;
                avgStencils[srcDev]->at(mapIdx) = 1;
                assert(srcIndex > 0 && srcIndex < avgPartition.getSize() * 4);
              }
              // Voxel can only be on one GPU...
              break;
            }
          }
          voxCounter++;
        }
  }
  assert(voxCounter == numAvgVoxels);

  // Create one CPU thread per GPU
#pragma omp parallel num_threads(nd)
  {
    std::stringstream ss;

    const int srcDev = omp_get_thread_num();
    CUDA_RT_CALL(hipSetDevice(srcDev));
    CUDA_RT_CALL(hipFree(0));

    SimulationParams* param = new SimulationParams(*cmptParams);
    m_params.at(srcDev) = param;
    SimulationState* state = new SimulationState();
    m_state.at(srcDev) = state;

    // Initialize distribution functions for temperature and velocity
    const Partition partition = getDevicePartition(srcDev);

    state->df = new DistributionFunction(19, nx, ny, nz, nd, partitioning);
    state->df_tmp = new DistributionFunction(19, nx, ny, nz, nd, partitioning);
    state->dfT = new DistributionFunction(7, nx, ny, nz, nd, partitioning);
    state->dfT_tmp = new DistributionFunction(7, nx, ny, nz, nd, partitioning);
    state->dfTeff = new DistributionFunction(1, nx, ny, nz, nd, partitioning);
    state->dfTeff_tmp =
        new DistributionFunction(1, nx, ny, nz, nd, partitioning);

    state->df->allocate(partition);
    state->df_tmp->allocate(partition);
    state->dfT->allocate(partition);
    state->dfT_tmp->allocate(partition);
    state->dfTeff->allocate(partition);
    state->dfTeff_tmp->allocate(partition);

    runInitKernel(state->df, state->dfT, partition, 1.0, 0, 0, 0, param->Tinit);
    runInitKernel(state->df_tmp, state->dfT_tmp, partition, 1.0, 0, 0, 0,
                  param->Tinit);
    state->dfTeff->fill(param->Tinit);
    state->dfTeff_tmp->fill(param->Tinit);
    ss << "Allocated partition " << partition << " on GPU" << srcDev
       << std::endl;

    // Arrays for averaging and plotting using back buffering
    const Partition partitionNoGhostLayer(
        partition.getMin(), partition.getMax(), vector3<int>(0, 0, 0));

    state->avg =
        new DistributionArray<real>(4, nx, ny, nz, nd, 0, partitioning);
    state->avg_tmp =
        new DistributionArray<real>(4, nx, ny, nz, nd, 0, partitioning);
    state->avg->allocate(partitionNoGhostLayer);
    state->avg_tmp->allocate(partitionNoGhostLayer);
    state->avg->fill(0);
    state->avg_tmp->fill(0);

    state->avgMap = new thrust::device_vector<int>(*avgMaps[srcDev]);
    state->avgStencil = new thrust::host_vector<int>(*avgStencils[srcDev]);

    state->avgResult =
        new DistributionArray<real>(4, numAvgVoxels, 0, 0, 1, 0, partitioning);
    state->avgResult->allocate();
    state->avgResult->fill(0);
    assert(state->avgResult->size(state->avgResult->getPartition()) ==
           4 * numAvgVoxels);

    // GPU local plot array with back buffering
    state->plot =
        new DistributionArray<real>(1, nx, ny, nz, nd, 0, partitioning);
    state->plot_tmp =
        new DistributionArray<real>(1, nx, ny, nz, nd, 0, partitioning);
    state->plot->allocate(partitionNoGhostLayer);
    state->plot_tmp->allocate(partitionNoGhostLayer);
    state->plot->fill(0);
    state->plot_tmp->fill(0);

    // Scatter voxel array into partitions
    state->voxels = new VoxelArray(nx, ny, nz, nd, partitioning);
    state->voxels->allocate(partitionNoGhostLayer);
    state->voxels->scatter(*voxels, partitionNoGhostLayer);

    // Upload boundary conditions array
    state->bcs = new thrust::device_vector<BoundaryCondition>(bcs->size());

    CUDA_RT_CALL(hipDeviceSynchronize());
    std::cout << ss.str();
  }  // end omp parallel

  std::cout << "LBM initialized" << std::endl;
}

void KernelInterface::uploadBCs(std::shared_ptr<BoundaryConditions> bcs) {
#pragma omp parallel num_threads(m_nd)
  {
    const int srcDev = omp_get_thread_num();
    CUDA_RT_CALL(hipSetDevice(srcDev));
    SimulationState* state = m_state.at(srcDev);
    *state->bcs = *bcs;
  }
}

void KernelInterface::getMinMax(real* min,
                                real* max,
                                thrust::host_vector<real>* histogram) {
  // *min = 20.0f;
  // *max = 30.0f;
  *min = REAL_MAX;
  *max = REAL_MIN;
  thrust::host_vector<real> mins(m_nd);
  thrust::host_vector<real> maxes(m_nd);
  thrust::fill(histogram->begin(), histogram->end(), 0.0);

#pragma omp parallel num_threads(m_nd)
  {
    const int srcDev = omp_get_thread_num();
    CUDA_RT_CALL(hipSetDevice(srcDev));
    const Partition partition = getDevicePartition(srcDev);
    const Partition partitionNoGhostLayer(
        partition.getMin(), partition.getMax(), vector3<int>(0, 0, 0));
    SimulationState* state = m_state.at(srcDev);
    mins[srcDev] = state->plot->getMin(partitionNoGhostLayer);
    maxes[srcDev] = state->plot->getMax(partitionNoGhostLayer);
#pragma omp barrier
#pragma omp single
    {
      *max = *thrust::max_element(maxes.begin(), maxes.end());
      *min = *thrust::min_element(mins.begin(), mins.end());
    }
    int nBins = histogram->size();
    thrust::host_vector<int> result(nBins);
    LatticeHistogram lHist;
    thrust::device_vector<real>* input =
        state->plot->getDeviceVector(partitionNoGhostLayer);
    lHist.calculate(input, *min, *max, nBins, &result);
#pragma omp critical
    for (int i = 0; i < nBins; i++) (*histogram)[i] += result[i];
  }
  for (int i = 0; i < histogram->size(); i++) (*histogram)[i] /= getSize();
}

void KernelInterface::resetDfs() {
#pragma omp parallel num_threads(m_nd)
  {
    const int srcDev = omp_get_thread_num();
    CUDA_RT_CALL(hipSetDevice(srcDev));
    const Partition partition = getDevicePartition(srcDev);
    SimulationParams* param = m_params.at(srcDev);
    SimulationState* state = m_state.at(srcDev);
    runInitKernel(state->df, state->dfT, partition, 1.0, 0, 0, 0, param->Tinit);
    runInitKernel(state->df_tmp, state->dfT_tmp, partition, 1.0, 0, 0, 0,
                  param->Tinit);
    state->dfTeff->fill(param->Tinit);
    state->dfTeff_tmp->fill(param->Tinit);
  }
}
