#include "hip/hip_runtime.h"
#include "KernelInterface.hpp"

void KernelInterface::runInitKernel(DistributionFunction* df,
                                    DistributionFunction* dfT,
                                    Partition partition,
                                    float rho,
                                    float vx,
                                    float vy,
                                    float vz,
                                    float T) {
  float sq_term = -1.5f * (vx * vx + vy * vy + vz * vz);
  vector3<size_t> n = partition.getArrayExtents();
  dim3 gridSize(n.y(), n.z(), 1);
  dim3 blockSize(n.x(), 1, 1);
  real_t* dfPtr = df->gpu_ptr(partition);
  real_t* dfTPtr = dfT->gpu_ptr(partition);

  InitKernel<<<gridSize, blockSize>>>(dfPtr, dfTPtr, n.x(), n.y(), n.z(), rho,
                                      vx, vy, vz, T, sq_term);
  CUDA_CHECK_ERRORS("InitKernel");
}

void KernelInterface::runComputeKernelInterior(
    const Partition partition,
    SimulationParams* param,
    SimulationState* state,
    DisplayQuantity::Enum displayQuantity,
    hipStream_t stream) {
  vector3<size_t> n = partition.getExtents() - partition.getGhostLayer() * (size_t)2;

  real_t* dfPtr = state->df->gpu_ptr(partition);
  real_t* df_tmpPtr = state->df_tmp->gpu_ptr(partition);
  real_t* dfTPtr = state->dfT->gpu_ptr(partition);
  real_t* dfT_tmpPtr = state->dfT_tmp->gpu_ptr(partition);
  real_t* dfTeffPtr = state->dfTeff->gpu_ptr(partition);
  real_t* dfTeff_tmpPtr = state->dfTeff_tmp->gpu_ptr(partition);

  Partition partitionNoGhostLayer(partition.getMin(), partition.getMax(),
                                  vector3<size_t>(0, 0, 0));
  real_t* avgSrcPtr = state->avg->gpu_ptr(partitionNoGhostLayer);
  real_t* avgDstPtr = state->avg_tmp->gpu_ptr(partitionNoGhostLayer);
  real_t* plotPtr = state->plot_tmp->gpu_ptr(partitionNoGhostLayer);
  voxel_t* voxelPtr = state->voxels->gpu_ptr(partitionNoGhostLayer);

  BoundaryCondition* bcsPtr = thrust::raw_pointer_cast(&(*state->bcs)[0]);

  dim3 gridSize(n.y(), n.z(), 1);
  dim3 blockSize(n.x(), 1, 1);
  if (m_method == LBM::BGK)
    ComputeKernel<LBM::BGK, D3Q4::ORIGIN><<<gridSize, blockSize, 0, stream>>>(
        partition, dfPtr, df_tmpPtr, dfTPtr, dfT_tmpPtr, dfTeffPtr,
        dfTeff_tmpPtr, voxelPtr, bcsPtr, m_dt, param->nu, param->C, param->nuT,
        param->Pr_t, param->gBetta, param->Tref, avgSrcPtr, avgDstPtr,
        displayQuantity, plotPtr);
  else if (m_method == LBM::MRT)
    ComputeKernel<LBM::MRT, D3Q4::ORIGIN><<<gridSize, blockSize, 0, stream>>>(
        partition, dfPtr, df_tmpPtr, dfTPtr, dfT_tmpPtr, dfTeffPtr,
        dfTeff_tmpPtr, voxelPtr, bcsPtr, m_dt, param->nu, param->C, param->nuT,
        param->Pr_t, param->gBetta, param->Tref, avgSrcPtr, avgDstPtr,
        displayQuantity, plotPtr);
  CUDA_CHECK_ERRORS("ComputeKernelInterior");
}

void KernelInterface::runComputeKernelBoundary(
    D3Q4::Enum direction,
    const Partition partition,
    SimulationParams* param,
    SimulationState* state,
    DisplayQuantity::Enum displayQuantity,
    hipStream_t stream) {
  vector3<size_t> n = partition.getExtents();

  real_t* dfPtr = state->df->gpu_ptr(partition);
  real_t* df_tmpPtr = state->df_tmp->gpu_ptr(partition);
  real_t* dfTPtr = state->dfT->gpu_ptr(partition);
  real_t* dfT_tmpPtr = state->dfT_tmp->gpu_ptr(partition);
  real_t* dfTeffPtr = state->dfTeff->gpu_ptr(partition);
  real_t* dfTeff_tmpPtr = state->dfTeff_tmp->gpu_ptr(partition);

  Partition partitionNoGhostLayer(partition.getMin(), partition.getMax(),
                                  vector3<size_t>(0, 0, 0));
  real_t* avgSrcPtr = state->avg->gpu_ptr(partitionNoGhostLayer);
  real_t* avgDstPtr = state->avg_tmp->gpu_ptr(partitionNoGhostLayer);
  real_t* plotPtr = state->plot_tmp->gpu_ptr(partitionNoGhostLayer);
  voxel_t* voxelPtr = state->voxels->gpu_ptr(partitionNoGhostLayer);

  BoundaryCondition* bcsPtr = thrust::raw_pointer_cast(&(*state->bcs)[0]);

  if (direction == D3Q4::X_AXIS) {
    dim3 gridSize(n.z(), 2, 1);
    dim3 blockSize(n.y(), 1, 1);
    if (m_method == LBM::BGK)
      ComputeKernel<LBM::BGK, D3Q4::X_AXIS><<<gridSize, blockSize, 0, stream>>>(
          partition, dfPtr, df_tmpPtr, dfTPtr, dfT_tmpPtr, dfTeffPtr,
          dfTeff_tmpPtr, voxelPtr, bcsPtr, m_dt, param->nu, param->C,
          param->nuT, param->Pr_t, param->gBetta, param->Tref, avgSrcPtr,
          avgDstPtr, displayQuantity, plotPtr);
    else if (m_method == LBM::MRT)
      ComputeKernel<LBM::MRT, D3Q4::X_AXIS><<<gridSize, blockSize, 0, stream>>>(
          partition, dfPtr, df_tmpPtr, dfTPtr, dfT_tmpPtr, dfTeffPtr,
          dfTeff_tmpPtr, voxelPtr, bcsPtr, m_dt, param->nu, param->C,
          param->nuT, param->Pr_t, param->gBetta, param->Tref, avgSrcPtr,
          avgDstPtr, displayQuantity, plotPtr);
    CUDA_CHECK_ERRORS("ComputeKernelBoundaryX");
  }
  if (direction == D3Q4::Y_AXIS) {
    dim3 gridSize(n.z(), 2, 1);
    dim3 blockSize(n.x(), 1, 1);
    if (m_method == LBM::BGK)
      ComputeKernel<LBM::BGK, D3Q4::Y_AXIS><<<gridSize, blockSize, 0, stream>>>(
          partition, dfPtr, df_tmpPtr, dfTPtr, dfT_tmpPtr, dfTeffPtr,
          dfTeff_tmpPtr, voxelPtr, bcsPtr, m_dt, param->nu, param->C,
          param->nuT, param->Pr_t, param->gBetta, param->Tref, avgSrcPtr,
          avgDstPtr, displayQuantity, plotPtr);
    else if (m_method == LBM::MRT)
      ComputeKernel<LBM::MRT, D3Q4::Y_AXIS><<<gridSize, blockSize, 0, stream>>>(
          partition, dfPtr, df_tmpPtr, dfTPtr, dfT_tmpPtr, dfTeffPtr,
          dfTeff_tmpPtr, voxelPtr, bcsPtr, m_dt, param->nu, param->C,
          param->nuT, param->Pr_t, param->gBetta, param->Tref, avgSrcPtr,
          avgDstPtr, displayQuantity, plotPtr);
    CUDA_CHECK_ERRORS("ComputeKernelBoundaryY");
  }
  if (direction == D3Q4::Z_AXIS) {
    dim3 gridSize(n.y(), 2, 1);
    dim3 blockSize(n.x(), 1, 1);
    if (m_method == LBM::BGK)
      ComputeKernel<LBM::BGK, D3Q4::Z_AXIS><<<gridSize, blockSize, 0, stream>>>(
          partition, dfPtr, df_tmpPtr, dfTPtr, dfT_tmpPtr, dfTeffPtr,
          dfTeff_tmpPtr, voxelPtr, bcsPtr, m_dt, param->nu, param->C,
          param->nuT, param->Pr_t, param->gBetta, param->Tref, avgSrcPtr,
          avgDstPtr, displayQuantity, plotPtr);
    else if (m_method == LBM::MRT)
      ComputeKernel<LBM::MRT, D3Q4::Z_AXIS><<<gridSize, blockSize, 0, stream>>>(
          partition, dfPtr, df_tmpPtr, dfTPtr, dfT_tmpPtr, dfTeffPtr,
          dfTeff_tmpPtr, voxelPtr, bcsPtr, m_dt, param->nu, param->C,
          param->nuT, param->Pr_t, param->gBetta, param->Tref, avgSrcPtr,
          avgDstPtr, displayQuantity, plotPtr);
    CUDA_CHECK_ERRORS("ComputeKernelBoundaryZ");
  }
}

std::vector<hipStream_t> KernelInterface::exchange(unsigned int srcDev,
                                                    Partition partition,
                                                    D3Q7::Enum direction) {
  SimulationState* state = m_state.at(srcDev);
  Partition neighbour = state->df_tmp->getNeighbour(partition, direction);
  unsigned int dstDev = getPartitionDevice(neighbour);
  hipStream_t dfStream = getDfGhostLayerStream(srcDev, dstDev);
  hipStream_t dfTStream = getDfTGhostLayerStream(srcDev, dstDev);
  state->df_tmp->exchange(partition, m_state.at(dstDev)->df_tmp, neighbour,
                          direction, dfStream);
  state->dfT_tmp->exchange(partition, m_state.at(dstDev)->dfT_tmp, neighbour,
                           direction, dfTStream);
  state->dfTeff_tmp->exchange(partition, m_state.at(dstDev)->dfTeff_tmp,
                              neighbour, direction, dfTStream);
  CUDA_RT_CALL(hipStreamSynchronize(dfStream));
  CUDA_RT_CALL(hipStreamSynchronize(dfTStream));
  return std::vector<hipStream_t>{dfStream, dfTStream};
}

void KernelInterface::calculateAverages() {
  thrust::host_vector<real_t>* avgs =
      m_avgs->getHostVector(m_avgs->getPartition());
  for (size_t srcDev = 0; srcDev < m_nd; srcDev++) {
    SimulationState* state = m_state.at(srcDev);
    thrust::host_vector<real_t> avgPartial =
        *state->avgResult->getHostVector(state->avgResult->getPartition());
    thrust::host_vector<int> avgStencil = *state->avgStencil;
    thrust::counting_iterator<int> iter(0);

    thrust::gather_if(iter, iter + avgPartial.size(), avgStencil.begin(),
                      avgPartial.begin(), avgs->begin());
  }
  m_avgs->upload();
}

LatticeAverage KernelInterface::getAverage(VoxelCuboid vol,
                                           uint64_t deltaTicks) {
  unsigned int offset = m_avgOffsets[vol];
  unsigned int size = vol.getNumVoxels();
  real_t ticks = static_cast<real_t>(deltaTicks);
  Partition partition = m_avgs->getPartition();
  real_t temperature = m_avgs->getAverage(partition, 0, offset, size, ticks);
  real_t velocityX = m_avgs->getAverage(partition, 1, offset, size, ticks);
  real_t velocityY = m_avgs->getAverage(partition, 2, offset, size, ticks);
  real_t velocityZ = m_avgs->getAverage(partition, 3, offset, size, ticks);
  return LatticeAverage(temperature, velocityX, velocityY, velocityZ);
}

void KernelInterface::compute(DisplayQuantity::Enum displayQuantity,
                              vector3<int> slicePos,
                              real_t* sliceX,
                              real_t* sliceY,
                              real_t* sliceZ,
                              bool runSimulation) {
#pragma omp parallel num_threads(m_nd)
  {
    const int srcDev = omp_get_thread_num() % m_nd;

    CUDA_RT_CALL(hipSetDevice(srcDev));

    SimulationParams* param = m_params.at(srcDev);
    SimulationState* state = m_state.at(srcDev);
    const Partition partition = getDevicePartition(srcDev);
    const Partition partitionNoGhostLayer(
        partition.getMin(), partition.getMax(), vector3<int>(0, 0, 0));

    const hipStream_t plotStream = getPlotStream(srcDev);
    const hipStream_t computeStream = getComputeStream(srcDev);
    const hipStream_t computeBoundaryStream = getComputeBoundaryStream(srcDev);
    const hipStream_t avgStream = getAvgStream(srcDev);

    // Compute LBM lattice boundary sites
    if (partition.getGhostLayer().x() > 0 && runSimulation) {
      runComputeKernelBoundary(D3Q4::X_AXIS, partition, param, state,
                               displayQuantity, computeBoundaryStream);
    }
    if (partition.getGhostLayer().y() > 0 && runSimulation) {
      runComputeKernelBoundary(D3Q4::Y_AXIS, partition, param, state,
                               displayQuantity, computeBoundaryStream);
    }
    if (partition.getGhostLayer().z() > 0 && runSimulation) {
      runComputeKernelBoundary(D3Q4::Z_AXIS, partition, param, state,
                               displayQuantity, computeBoundaryStream);
    }

    // Compute inner lattice sites (excluding boundaries)
    if (runSimulation)
      runComputeKernelInterior(partition, param, state, displayQuantity,
                               computeStream);

    // Gather the plot to draw the display slices
    if (slicePos != vector3<int>(-1, -1, -1)) {
      state->plot->gatherSlice(slicePos, 0, 0, partitionNoGhostLayer, m_plot,
                               plotStream);
    }

    // Gather averages from GPU array
    if (runSimulation) {
      thrust::device_vector<real_t>* input =
          state->avg->getDeviceVector(partitionNoGhostLayer);
      thrust::device_vector<real_t>* output =
          state->avgResult->getDeviceVector(state->avgResult->getPartition());
      thrust::gather(thrust::cuda::par.on(avgStream), state->avgMap->begin(),
                     state->avgMap->end(), input->begin(), output->begin());
      state->avgResult->download();
      if (m_resetAvg) {
        state->avg->fill(0, avgStream);
        state->avg_tmp->fill(0, avgStream);
      }
    }

    // Wait for boundary lattice sites to finish computing
    CUDA_RT_CALL(hipStreamSynchronize(computeBoundaryStream));

    // Perform ghost layer exchanges
    if (partition.getGhostLayer().x() > 0 && runSimulation) {
      std::vector<hipStream_t> streamsPos =
          exchange(srcDev, partition, D3Q7::X_AXIS_POS);
      std::vector<hipStream_t> streamsNeg =
          exchange(srcDev, partition, D3Q7::X_AXIS_NEG);
      for (hipStream_t stream : streamsPos)
        CUDA_RT_CALL(hipStreamSynchronize(stream));
      for (hipStream_t stream : streamsNeg)
        CUDA_RT_CALL(hipStreamSynchronize(stream));
    }

#pragma omp barrier
    if (partition.getGhostLayer().y() > 0 && runSimulation) {
      std::vector<hipStream_t> streamsPos =
          exchange(srcDev, partition, D3Q7::Y_AXIS_POS);
      std::vector<hipStream_t> streamsNeg =
          exchange(srcDev, partition, D3Q7::Y_AXIS_NEG);
      for (hipStream_t stream : streamsPos)
        CUDA_RT_CALL(hipStreamSynchronize(stream));
      for (hipStream_t stream : streamsNeg)
        CUDA_RT_CALL(hipStreamSynchronize(stream));
    }

#pragma omp barrier
    if (partition.getGhostLayer().z() > 0 && runSimulation) {
      std::vector<hipStream_t> streamsPos =
          exchange(srcDev, partition, D3Q7::Z_AXIS_POS);
      std::vector<hipStream_t> streamsNeg =
          exchange(srcDev, partition, D3Q7::Z_AXIS_NEG);
      for (hipStream_t stream : streamsPos)
        CUDA_RT_CALL(hipStreamSynchronize(stream));
      for (hipStream_t stream : streamsNeg)
        CUDA_RT_CALL(hipStreamSynchronize(stream));
    }
    CUDA_RT_CALL(hipStreamSynchronize(plotStream));

#pragma omp barrier
    if (srcDev == 0 && slicePos != vector3<int>(-1, -1, -1)) {
      real_t* plot3dPtr = m_plot->gpu_ptr(m_plot->getPartition());
      dim3 blockSize, gridSize;

      vector3<size_t> n = getExtents();
      setExtents(n.y() * n.z(), BLOCK_SIZE_DEFAULT, &blockSize, &gridSize);
      SliceXRenderKernel<<<gridSize, blockSize, 0, plotStream>>>(
          plot3dPtr, n.x(), n.y(), n.z(), sliceX, slicePos.x());
      CUDA_CHECK_ERRORS("SliceXRenderKernel");

      setExtents(n.x() * n.z(), BLOCK_SIZE_DEFAULT, &blockSize, &gridSize);
      SliceYRenderKernel<<<gridSize, blockSize, 0, plotStream>>>(
          plot3dPtr, n.x(), n.y(), n.z(), sliceY, slicePos.y());
      CUDA_CHECK_ERRORS("SliceYRenderKernel");

      setExtents(n.x() * n.y(), BLOCK_SIZE_DEFAULT, &blockSize, &gridSize);
      SliceZRenderKernel<<<gridSize, blockSize, 0, plotStream>>>(
          plot3dPtr, n.x(), n.y(), n.z(), sliceZ, slicePos.z());
      CUDA_CHECK_ERRORS("SliceZRenderKernel");
    }

    CUDA_RT_CALL(hipStreamSynchronize(computeStream));
    CUDA_RT_CALL(hipStreamSynchronize(avgStream));
    CUDA_RT_CALL(hipStreamSynchronize(plotStream));

#pragma omp barrier
    if (runSimulation) {
      DistributionFunction::swap(state->df, state->df_tmp);
      DistributionFunction::swap(state->dfT, state->dfT_tmp);
      DistributionFunction::swap(state->dfTeff, state->dfTeff_tmp);
      DistributionFunction::swap(state->plot, state->plot_tmp);
      DistributionFunction::swap(state->avg, state->avg_tmp);
    }
  }
  m_resetAvg = false;
}

KernelInterface::KernelInterface(
    const size_t nx,
    const size_t ny,
    const size_t nz,
    const real_t dt,
    const std::shared_ptr<SimulationParams> cmptParams,
    const std::shared_ptr<BoundaryConditions> bcs,
    const std::shared_ptr<VoxelArray> voxels,
    const std::shared_ptr<VoxelCuboidArray> avgVols,
    const size_t nd,
    const LBM::Enum method,
    const D3Q4::Enum partitioning)
    : P2PLattice(nx, ny, nz, nd, partitioning),
      m_params(nd),
      m_state(nd),
      m_method(method),
      m_resetAvg(false),
      m_dt(dt) {
  std::cout << "Initializing LBM data structures..." << std::endl;
  CUDA_RT_CALL(hipSetDevice(0));
  CUDA_RT_CALL(hipFree(0));

  // Arrays for gathering distributed plot with back buffering
  m_plot = new DistributionArray<real_t>(1, nx, ny, nz, 1, 0, partitioning);
  m_plot_tmp = new DistributionArray<real_t>(1, nx, ny, nz, 1, 0, partitioning);
  m_plot->allocate();
  m_plot_tmp->allocate();
  m_plot->fill(0);
  m_plot_tmp->fill(0);

  // Array for gathering simulation averages
  unsigned int numAvgVoxels = 0;
  for (size_t i = 0; i < avgVols->size(); i++) {
    VoxelCuboid vol = avgVols->at(i);
    m_avgOffsets[vol] = numAvgVoxels;
    vector3<int> ext = vol.getExtents();
    numAvgVoxels += ext.x() * ext.y() * ext.z();
  }
  m_avgs =
      new DistributionArray<real_t>(4, numAvgVoxels, 0, 0, 1, 0, partitioning);
  m_avgs->allocate();
  m_avgs->fill(0);

  // Create maps and stencils for averaging with gather_if
  std::vector<int>* avgMaps[nd];
  std::vector<int>* avgStencils[nd];
  for (size_t srcDev = 0; srcDev < nd; srcDev++) {
    avgMaps[srcDev] = new std::vector<int>(4 * numAvgVoxels, 0);
    avgStencils[srcDev] = new std::vector<int>(4 * numAvgVoxels, 0);
  }
  int voxCounter = 0;
  // Loop over all volumes
  for (size_t i = 0; i < avgVols->size(); i++) {
    VoxelCuboid avg = avgVols->at(i);
    // Global minimum and maximum of volumes
    vector3<int> aMin = avg.getMin();
    vector3<int> aMax = avg.getMax();

    // Loop over all voxels in volume
    for (int z = aMin.z(); z < aMax.z(); z++)
      for (int y = aMin.y(); y < aMax.y(); y++)
        for (int x = aMin.x(); x < aMax.x(); x++) {
          // Voxel in volume in global coordinates
          vector3<unsigned int> vox(x, y, z);
          // Loop over all lattice partitions
          for (size_t srcDev = 0; srcDev < nd; srcDev++) {
            const Partition latticePartition = getDevicePartition(srcDev);
            const Partition avgPartition(latticePartition.getMin(),
                                         latticePartition.getMax(),
                                         vector3<size_t>(0, 0, 0));

            const vector3<unsigned int> pMin = avgPartition.getMin();
            const vector3<unsigned int> pMax = avgPartition.getMax();
            const vector3<size_t> pExt = avgPartition.getExtents();

            // Check if voxel is inside partition
            if ((pMin.x() <= vox.x() && vox.x() < pMax.x()) &&
                (pMin.y() <= vox.y() && vox.y() < pMax.y()) &&
                (pMin.z() <= vox.z() && vox.z() < pMax.z())) {
              // Convert voxel to local coordinate in partition
              vector3<unsigned int> srcPos = vox - pMin;
              // Loop over temperature (0) and each velocity (1-3)
              for (int q = 0; q < 4; q++) {
                // Convert local coordinate to array index
                int srcIndex = I4D(q, srcPos.x(), srcPos.y(), srcPos.z(),
                                   pExt.x(), pExt.y(), pExt.z());
                int mapIdx = q * numAvgVoxels + voxCounter;
                avgMaps[srcDev]->at(mapIdx) = srcIndex;
                avgStencils[srcDev]->at(mapIdx) = 1;
                assert(srcIndex > 0 && srcIndex < avgPartition.getSize() * 4);
              }
              // Voxel can only be on one GPU...
              break;
            }
          }
          voxCounter++;
        }
  }
  assert(voxCounter == numAvgVoxels);

  // Create one CPU thread per GPU
#pragma omp parallel num_threads(nd)
  {
    std::stringstream ss;

    const int srcDev = omp_get_thread_num();
    CUDA_RT_CALL(hipSetDevice(srcDev));
    CUDA_RT_CALL(hipFree(0));

    SimulationParams* param = new SimulationParams(*cmptParams);
    m_params.at(srcDev) = param;
    SimulationState* state = new SimulationState();
    m_state.at(srcDev) = state;

    // Initialize distribution functions for temperature and velocity
    const Partition partition = getDevicePartition(srcDev);

    state->df = new DistributionFunction(19, nx, ny, nz, nd, partitioning);
    state->df_tmp = new DistributionFunction(19, nx, ny, nz, nd, partitioning);
    state->dfT = new DistributionFunction(7, nx, ny, nz, nd, partitioning);
    state->dfT_tmp = new DistributionFunction(7, nx, ny, nz, nd, partitioning);
    state->dfTeff = new DistributionFunction(1, nx, ny, nz, nd, partitioning);
    state->dfTeff_tmp =
        new DistributionFunction(1, nx, ny, nz, nd, partitioning);

    state->df->allocate(partition);
    state->df_tmp->allocate(partition);
    state->dfT->allocate(partition);
    state->dfT_tmp->allocate(partition);
    state->dfTeff->allocate(partition);
    state->dfTeff_tmp->allocate(partition);

    runInitKernel(state->df, state->dfT, partition, 1.0, 0, 0, 0, param->Tinit);
    runInitKernel(state->df_tmp, state->dfT_tmp, partition, 1.0, 0, 0, 0,
                  param->Tinit);
    state->dfTeff->fill(param->Tinit);
    state->dfTeff_tmp->fill(param->Tinit);
    ss << "Allocated partition " << partition << " on GPU" << srcDev
       << std::endl;

    // Arrays for averaging and plotting using back buffering
    const Partition partitionNoGhostLayer(
        partition.getMin(), partition.getMax(), vector3<int>(0, 0, 0));

    state->avg =
        new DistributionArray<real_t>(4, nx, ny, nz, nd, 0, partitioning);
    state->avg_tmp =
        new DistributionArray<real_t>(4, nx, ny, nz, nd, 0, partitioning);
    state->avg->allocate(partitionNoGhostLayer);
    state->avg_tmp->allocate(partitionNoGhostLayer);
    state->avg->fill(0);
    state->avg_tmp->fill(0);

    state->avgMap = new thrust::device_vector<int>(*avgMaps[srcDev]);
    state->avgStencil = new thrust::host_vector<int>(*avgStencils[srcDev]);

    state->avgResult =
        new DistributionArray<real_t>(4, numAvgVoxels, 0, 0, 1, 0, partitioning);
    state->avgResult->allocate();
    state->avgResult->fill(0);
    assert(state->avgResult->size(state->avgResult->getPartition()) ==
           4 * numAvgVoxels);

    // GPU local plot array with back buffering
    state->plot =
        new DistributionArray<real_t>(1, nx, ny, nz, nd, 0, partitioning);
    state->plot_tmp =
        new DistributionArray<real_t>(1, nx, ny, nz, nd, 0, partitioning);
    state->plot->allocate(partitionNoGhostLayer);
    state->plot_tmp->allocate(partitionNoGhostLayer);
    state->plot->fill(0);
    state->plot_tmp->fill(0);

    // Scatter voxel array into partitions
    state->voxels = new VoxelArray(nx, ny, nz, nd, partitioning);
    state->voxels->allocate(partitionNoGhostLayer);
    state->voxels->scatter(*voxels, partitionNoGhostLayer);

    // Upload boundary conditions array
    state->bcs = new thrust::device_vector<BoundaryCondition>();
    state->bcs->insert(state->bcs->begin(), bcs->begin(), bcs->end());

    CUDA_RT_CALL(hipDeviceSynchronize());
    std::cout << ss.str();
  }  // end omp parallel

  std::cout << "LBM initialized" << std::endl;
}

void KernelInterface::uploadBCs(std::shared_ptr<BoundaryConditions> bcs) {
#pragma omp parallel num_threads(m_nd)
  {
    const int srcDev = omp_get_thread_num();
    CUDA_RT_CALL(hipSetDevice(srcDev));
    SimulationState* state = m_state.at(srcDev);
    thrust::copy(bcs->begin(), bcs->end(), state->bcs->begin());
  }
}

void KernelInterface::getMinMax(real_t* min,
                                real_t* max,
                                thrust::host_vector<real_t>* histogram) {
  // *min = 20.0f;
  // *max = 30.0f;
  *min = REAL_MAX;
  *max = REAL_MIN;
  thrust::host_vector<real_t> mins(m_nd);
  thrust::host_vector<real_t> maxes(m_nd);
  thrust::fill(histogram->begin(), histogram->end(), 0.0);

#pragma omp parallel num_threads(m_nd)
  {
    const int srcDev = omp_get_thread_num();
    CUDA_RT_CALL(hipSetDevice(srcDev));
    const Partition partition = getDevicePartition(srcDev);
    const Partition partitionNoGhostLayer(
        partition.getMin(), partition.getMax(), vector3<int>(0, 0, 0));
    SimulationState* state = m_state.at(srcDev);
    mins[srcDev] = state->plot->getMin(partitionNoGhostLayer);
    maxes[srcDev] = state->plot->getMax(partitionNoGhostLayer);
#pragma omp barrier
#pragma omp single
    {
      *max = *thrust::max_element(maxes.begin(), maxes.end());
      *min = *thrust::min_element(mins.begin(), mins.end());
    }
    int nBins = histogram->size();
    thrust::host_vector<int> result(nBins);
    LatticeHistogram lHist;
    thrust::device_vector<real_t>* input =
        state->plot->getDeviceVector(partitionNoGhostLayer);
    lHist.calculate(input, *min, *max, nBins, &result);
#pragma omp critical
    for (int i = 0; i < nBins; i++) (*histogram)[i] += result[i];
  }
  for (size_t i = 0; i < histogram->size(); i++) (*histogram)[i] /= getSize();
}

void KernelInterface::resetDfs() {
#pragma omp parallel num_threads(m_nd)
  {
    const int srcDev = omp_get_thread_num();
    CUDA_RT_CALL(hipSetDevice(srcDev));
    const Partition partition = getDevicePartition(srcDev);
    SimulationParams* param = m_params.at(srcDev);
    SimulationState* state = m_state.at(srcDev);
    runInitKernel(state->df, state->dfT, partition, 1.0, 0, 0, 0, param->Tinit);
    runInitKernel(state->df_tmp, state->dfT_tmp, partition, 1.0, 0, 0, 0,
                  param->Tinit);
    state->dfTeff->fill(param->Tinit);
    state->dfTeff_tmp->fill(param->Tinit);
  }
}
