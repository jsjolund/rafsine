#include "hip/hip_runtime.h"
#include "GatherKernel.hpp"

__global__ void GatherKernel(int* map,
                             int size,
                             int* stencil,
                             real_t* input,
                             real_t* output) {
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  if (i >= size) return;
  if (stencil[i]) output[i] = input[map[i]];
}
